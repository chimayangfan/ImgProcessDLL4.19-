#include "hip/hip_runtime.h"
#include"Imgsimulation.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
#include <string.h>
#include <fstream> 
#include <string>
#include <io.h>
#include <vector>
#include <time.h>  

#include <stdio.h>  
#include<algorithm>

#include "Thread.h"
#include "ThreadPoolExecutor.h"
#include"hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>//������
#include <Windows.h>
#include <GdiPlus.h>

#include <helper_string.h>
#include <npp.h>

#pragma comment( lib, "GdiPlus.lib" )
using namespace Gdiplus;
using namespace std;
using namespace cv;

//�����豸���ܶ���
#define ExtractPointThreads 1
#define CUDAStreams 2
int gHostImgblock = ExtractPointThreads * CUDAStreams;
int gDeviceCount;
int gHostPathImgNumber;
//����ͼƬ��С����block��thread���� 
Parameter gStructVarible{NULL,NULL,NULL,8,5120,5120,5120,60,30,300,640,640,0,9999,1,false};
Infomation SignPoint;

#define Pretreatment
	#ifdef Pretreatment
	#define ReadImageNumber 250
#endif // Pretreatment
unsigned char* gHostImage[250] = { NULL };
unsigned char* gHostColorImage[250] = { NULL };

unsigned char* rhost_in[CUDAStreams];//ҳ�����ڴ�
unsigned char* rDev_in[CUDAStreams];//�豸�ڴ�
unsigned char* rDev_padding[CUDAStreams];//���߽���ͼ���ڴ�   qwt7.26
unsigned char* rgpu_2val[CUDAStreams];//��ֵ��ͼ
unsigned char* rgpu_counter[CUDAStreams];//����ͼ����ִ��findcountores֮�������

hipStream_t *rcS;


//-------------------------��λ��Model����-----------------------------//
typedef struct
{
	short RecXmin;
	short RecYmin;
	short RecXmax;
	short RecYmax;
}RecData;//��λ�����ݽṹ
vector<RecData> gHostRecData;//CPU��λ����������
int gRectRealNum;//��λ�е�ʵ������
//��λ�и�������
int gImgXcenter;//ͼƬ�Ҷ����ĵ㣨�����õĻҶ����ļ�Ȩƽ����
int gImgYcenter;
int gXcenterOffset;//��Χ��ƫ����(��Χ�и���ʱ���õ�ֵ)
int gYcenterOffset;
//��������
struct CircleInfo
{
	short index;
	short length;
	short area;
	short xpos;
	short ypos;
};
//-------------------------------------------------------����----------------------------------------//

/*------------------------------------------------�˺���--------------------------------------------------*/
//--------------------------------------------------------��ʼ---------------------------------------------//

/*���ͼ��߽�*/
//����Ϊԭͼͼ��ͼ��߶ȡ�ͼ����  ���Ϊ����Ŀ��  ����Ŀ�ȼ��㹫ʽ   int imgWidth = (width + 127) / 128 * 128;
__global__ void  CopyMakeBorder(const unsigned char *src, unsigned char *dst, Parameter devpar)
{
	const int Id_y = threadIdx.x + blockIdx.x*blockDim.x;//Id_y��ʾͼ��������
	const int Id_x = blockIdx.y;
	if (Id_y <  devpar.ImgWidth)
	{
		dst[Id_y + Id_x * devpar.ImgMakeborderWidth] = src[Id_y + Id_x * devpar.ImgWidth];
	}
}

/*��ֵ��*/
__global__ void Binarization(unsigned char *psrcgray, unsigned char *pdst2val, unsigned char *pdstcounter, Parameter devpar)
{
	const int Id = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;//����άgrid��һάblock��
	int temp = int(psrcgray[Id]);//�Ĵ����������أ���߷ô�Ч��								
	if (Id < devpar.ImgMakeborderWidth * devpar.ImgHeight*devpar.PictureNum)
	{
		pdst2val[Id] = unsigned char(255 * int(temp>devpar.Threshold));//��ֵ�������ü�������֧�ṹ
		pdstcounter[Id] = unsigned char(255 * int(temp>devpar.Threshold));
	}
}

/*��ȡ��������Ե��⣩*/
//����
__global__  void Dilation(unsigned char *psrc, unsigned char *pdst, Parameter devpar)
{
	const int Id_y = threadIdx.x + blockIdx.x *blockDim.x;//Id_x��������Ϣ  Id_y��������Ϣ
	const int Id_x = blockIdx.y;
	int temp;
	if (Id_y> 0 && Id_y < (devpar.ImgMakeborderWidth - 1) && Id_x>0 && Id_x <devpar.ImgHeight*devpar.PictureNum - 1)
	{
		if (psrc[Id_y + Id_x * devpar.ImgMakeborderWidth] == 0)
		{
			temp = int(psrc[Id_y - 1 + (Id_x - 1)* devpar.ImgMakeborderWidth]) + int(psrc[Id_y + (Id_x - 1)* devpar.ImgMakeborderWidth]) + int(psrc[Id_y + 1 + (Id_x - 1)* devpar.ImgMakeborderWidth])
				+ int(psrc[Id_y - 1 + Id_x * devpar.ImgMakeborderWidth]) + int(psrc[Id_y + 1 + Id_x * devpar.ImgMakeborderWidth]) +
				int(psrc[Id_y - 1 + (Id_x + 1)* devpar.ImgMakeborderWidth]) + int(psrc[Id_y + (Id_x + 1)* devpar.ImgMakeborderWidth]) + int(psrc[Id_y + 1 + (Id_x + 1)* devpar.ImgMakeborderWidth]);
			pdst[Id_y + Id_x * devpar.ImgMakeborderWidth] = temp > 0 ? 255 : 0;
		}
	}

}

//��ʴ
__global__  void Erosion(unsigned char *psrc, unsigned char *pdst, Parameter devpar)
{
	const int Id_y = threadIdx.x + blockIdx.x *blockDim.x;//Id_y��������Ϣ  Id_x��������Ϣ
	const int Id_x = blockIdx.y;//Id_x��������Ϣ
	int temp;
	//����4����ֵ�Ϳ��ڲ��㣬��ȡ������Ϣ�����ڵ�dst���Ǵ洢��������Ϣ
	if (Id_y > 0 && Id_y < (devpar.ImgMakeborderWidth - 1) && Id_x>0 && Id_x <devpar.ImgHeight*devpar.PictureNum - 1)
	{
		if (psrc[Id_y + Id_x * devpar.ImgMakeborderWidth] != 0)
		{
			temp = int(psrc[Id_y + (Id_x - 1)*devpar.ImgMakeborderWidth]) + int(psrc[Id_y - 1 + Id_x * devpar.ImgMakeborderWidth]) +
				int(psrc[Id_y + 1 + Id_x * devpar.ImgMakeborderWidth]) + int(psrc[Id_y + (Id_x + 1)*devpar.ImgMakeborderWidth]);//��4����ʴ
			pdst[Id_y + Id_x * devpar.ImgMakeborderWidth] = temp >= 1020 ? 0 : 255;
		}
	}
}

/*������ȡ*/
//srcΪ�������飨��Ե���������c_length��ȡ���ܳ�ֵ��   (x_min,y_min)��(x_max,y_max)���ڱ���������������,����x�йص�Ϊ��������y�йصı���������
__global__  void GetCounter(unsigned char *src, short *c_length, short* x_min, short * y_min, short* x_max, short *y_max, Parameter devpar)
{
	//�����������飬���ڸ���������,��ʼ������Ϊ���ҷ���0��λ����˳ʱ����ת45�㣨������1��
	const  int direction_y[8] = { 1,1,0,-1,-1,-1,0,1 };
	const  int direction_x[8] = { 0,1,1,1,0,-1,-1,-1 };
	//��ȡ����������
	const int y = (blockIdx.x*blockDim.x + threadIdx.x) * 8;//y��������
	const int x = blockIdx.y * 8;//x��������
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	/*��ʼ��������ֵ*/
	c_length[Id] = 0;
	x_min[Id] = 0;
	x_max[Id] = 0;
	y_min[Id] = 0;
	y_max[Id] = 0;
	/*��ʼ������������Ϣ*/
	short x_pos_max = 0, x_pos_min = 0;
	short y_pos_max = 0, y_pos_min = 0;//����������������λ����Ϣ
	short  Point_counts = 0;//��������

	/*ѭ����ȡ�����ܳ���Ϣ*/
	if ((y / 8) <= (devpar.ImgWidth / 8))  //qwt815
	{
		for (int i = x; i < (x + 8); i++)
		{
			for (int j = y; j < (y + 8); j++)
			{
				if (255 == src[j + i * devpar.ImgMakeborderWidth])
				{
					//���±߽�����
					y_pos_max = j;
					y_pos_min = j;
					x_pos_max = i;
					x_pos_min = i;
					Point_counts = 1;//������Ŀ����ֵ
					// ��ʼ�㼰��ǰ��  
					short x_pos = i;//������
					short y_pos = j;//������
					bool first_time = false;//�Ƿ�ʱ��һ�λ�ȡ������
					short counts = 0;//����ѭ������
					short curr_d = 0;//������������������ȡֵ0-7��ʾ�������8�����õķ�λ
									 // ���и���  
					for (short cLengthCount = 0; cLengthCount < devpar.LengthMax; cLengthCount++)//�����ѭ��������Ҫ�ý������õ��ܳ����ֵ��ȷ��
					{
						//�������ǵ�
						short root_x = x_pos;
						short root_y = y_pos;
						//������������߽�����
						x_pos_max = x_pos_max > x_pos ? x_pos_max : x_pos;
						y_pos_max = y_pos_max > y_pos ? y_pos_max : y_pos;
						x_pos_min = x_pos_min < x_pos ? x_pos_min : x_pos;
						y_pos_min = y_pos_min < y_pos ? y_pos_min : y_pos;
						// ѭ���˴� :���ڻ�ȡ��һ��������
						for (counts = 0; counts < 8; counts++)
						{
							// ��ֹ��������  
							curr_d -= curr_d >= 8 ? 8 : 0;
							curr_d += curr_d < 0 ? 8 : 0;
							//��ʵ�ϣ�ֻ��Ҫ�ж�7�������ڵ���Ϣ(���˵�һ��֮��)����count=6ʱ�պ�ѭ������һ��������
							if (first_time && (counts == 6))//qwt 9.04 ��������˵�����һ�β��ܽ���if
							{
								continue;
							}
							//���±�ǵ�root;
							root_x = x_pos + direction_x[curr_d];//����������
							root_y = y_pos + direction_y[curr_d];//����������
							//�жϵ��Ƿ�Խ�磬����ͼ�����������
							if (root_x < 0 || root_x >= devpar.ImgHeight*devpar.PictureNum || root_y < 0 || root_y >= devpar.ImgWidth)
							{
								curr_d++;
								continue;
							}
							//������ڱ�Ե  
							if (255 == src[root_y + root_x * devpar.ImgMakeborderWidth])
							{
								curr_d -= 2;   //���µ�ǰ����  
								Point_counts++;
								//����b_pt:���ٵ�root��  
								x_pos = root_x;
								y_pos = root_y;
								break;   // ����forѭ��  
							}
							curr_d++;
						}   // end for  ��
							//���ٽ��������쳣����
						if (8 == counts || (x_pos >= (x + 8) && y_pos >= (y + 8)))
						{
							break;
						}
						//��������
						if (y_pos == j && x_pos == i)
						{
							//����������Ϣ
							c_length[Id] = Point_counts;
							x_min[Id] = x_pos_min;
							x_max[Id] = x_pos_max;
							y_min[Id] = y_pos_min;
							y_max[Id] = y_pos_max;
							break;
						}//��������if
						//�ж�
						if (cLengthCount == 0)
						{
							first_time = true;
						}
					}//��Χfor����			
				}//�ж�ǰ����if����
				j = y_pos_max > j ? y_pos_max : j;//���º�����������
			}//��һ��for����
			i = x_pos_max > i ? x_pos_max : i;
		}//�ڶ���for ����
	}
}//�˺�������

 /*���������ȡ*/
 //��������getCounter��ȡ�� x_min��x_max�����У� y_min��y_max�����С�
__global__  void GetInfo(unsigned char* src_gray, short *length, short* x_min, short * y_min, short* x_max, short *y_max, short *xpos, short*ypos, short *area, Parameter devpar)
{
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	short count = 0;//�����������
	int sum_gray = 0;//Բ������ĻҶ�ֵ֮��
	int x_sum = 0;//x�Ҷ�ֵ��Ȩ��
	int y_sum = 0;//y�Ҷ�ֵ��Ȩ��
	int mThreshold = devpar.Threshold;//��ֵ����ֵ
									  //���淽λ�б߽�
	short xmm = x_min[Id];
	short xmx = x_max[Id];
	short ymm = y_min[Id];
	short ymx = y_max[Id];
	short jcount = (ymx - ymm + 3) / 4 * 4;
	unsigned char temp0, temp1, temp2, temp3;//�üĴ����ݴ�ͼ�����ݣ���Сȫ���ڴ�ķ��ʣ���߷ô�Ч��
	area[Id] = 0;
	xpos[Id] = 0;
	ypos[Id] = 0;
	if (length[Id] > devpar.LengthMin)
	{
		//ѭ���Ż�,�������������һЩ�����ֵ����Ҫ����һ�£�
		for (int i = xmm; i <= xmx; i++)
			for (int j = ymm; j <= ymm + jcount; j = j + 4)
			{
				//��ֹԽ��
				temp0 = j > ymx ? 0 : 1;  //qwt
				temp1 = j + 1 > ymx ? 0 : 1;
				temp2 = j + 2 > ymx ? 0 : 1;
				temp3 = j + 3 > ymx ? 0 : 1;

				//���ݶ�ֵ����ֵ 
				temp0 *= src_gray[j   *temp0 + i * devpar.ImgMakeborderWidth] > mThreshold ? src_gray[j   *temp0 + i * devpar.ImgMakeborderWidth] : 0;
				temp1 *= src_gray[(j + 1)*temp1 + i * devpar.ImgMakeborderWidth] > mThreshold ? src_gray[(j + 1)*temp1 + i * devpar.ImgMakeborderWidth] : 0;
				temp2 *= src_gray[(j + 2)*temp2 + i * devpar.ImgMakeborderWidth] > mThreshold ? src_gray[(j + 2)*temp2 + i * devpar.ImgMakeborderWidth] : 0;
				temp3 *= src_gray[(j + 3)*temp3 + i * devpar.ImgMakeborderWidth] > mThreshold ? src_gray[(j + 3)*temp3 + i * devpar.ImgMakeborderWidth] : 0;

				count += temp0 > 0 ? 1 : 0; //�������
				count += temp1 > 0 ? 1 : 0;
				count += temp2 > 0 ? 1 : 0;
				count += temp3 > 0 ? 1 : 0;

				sum_gray += temp0 + temp1 + temp2 + temp3;


				x_sum += i * temp0 + i * temp1 + i * temp2 + i * temp3;
				y_sum += j * temp0 + (j + 1)*temp1 + (j + 2)*temp2 + (j + 3)*temp3;
			}
		area[Id] = count;
		xpos[Id] = x_sum / sum_gray;
		ypos[Id] = y_sum / sum_gray;
	}
}

//ɸѡ���ظ���Ϣ�ĺ���,����˺���Ҫ����ʧ��
__global__  void GetTrueInfo(short *xcenter, short *ycenter, short*index, short *sArea, Parameter devpar)
{
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	short temp = 0;
	index[Id] = 0;//����������
	if ((Id > devpar.ColThreadNum) && (Id < devpar.ColThreadNum*(devpar.RowThreadNum - 1)))
	{
		if (xcenter[Id] != 0)
		{
			//�ж�һ���̻߳�ȡ�������Ƿ���������ڵ��ҷ��̣߳���+1�����·��̣߳���+1����ȡ������һ�¡�����һ�����������ֵ
			//��
			temp += ((xcenter[Id] == xcenter[Id + 1]) && (ycenter[Id] == ycenter[Id + 1])) ? 1 : 0;//��
			temp += ((xcenter[Id] == xcenter[Id + devpar.ColThreadNum]) && (ycenter[Id] == ycenter[Id + devpar.ColThreadNum])) ? 1 : 0;//��
			temp += ((xcenter[Id] == xcenter[Id - devpar.ColThreadNum + 1]) && (ycenter[Id] == ycenter[Id - devpar.ColThreadNum + 1])) ? 1 : 0;//����
			temp += ((sArea[Id] > devpar.AreaMin) && (sArea[Id] < devpar.AreaMax)) ? 0 : 1;//qwt8-8
			index[Id] = temp > 0 ? 0 : Id;
		}
	}
}

/*����ģʽ��������ȡ*/
//���룺 ��λ��    �Ҷ�ͼ    ����ͼ
//����� �ܳ�  ���  ��������
__global__	void GetRecInfo(RecData* mRec, unsigned char *psrcgray, unsigned char *psrccounter,
	short *length, short* area, short *xpos, short *ypos, Parameter devpar)
{
	const int Id = threadIdx.x + blockIdx.x*blockDim.x;
	int mThreshold = devpar.Threshold;//��ֵ����ֵ
	short count = 0;//�����������
	int clengthCount = 0;
	short clength = 0;//�ܳ�����
	int sum_gray = 0;//Բ������ĻҶ�ֵ֮��
	int x_sum = 0;//x�Ҷ�ֵ��Ȩ��
	int y_sum = 0;//y�Ҷ�ֵ��Ȩ��

				  //��ȡ��λ��
	short xmm = mRec[Id].RecXmin;
	short xmx = mRec[Id].RecXmax;
	short ymm = mRec[Id].RecYmin;
	short ymx = mRec[Id].RecYmax;
	short jcount = (ymx - ymm + 3) / 4 * 4;//qwt
	unsigned char temp0, temp1, temp2, temp3;//temp���ڼ������ġ����
	unsigned char t0, t1, t2, t3;//t���ڼ����ܳ�
	area[Id] = 0;
	xpos[Id] = 0;
	ypos[Id] = 0;
	length[Id] = 0;
	//ѭ���Ż�,�������������һЩ�����ֵ����Ҫ����һ�£�
	for (int i = xmm; i <= xmx; i++)
		for (int j = ymm; j <= ymm + jcount; j = j + 4)
		{
			//��ֹԽ��
			temp0 = j    > ymx ? 0 : 1;
			temp1 = j + 1> ymx ? 0 : 1;
			temp2 = j + 2> ymx ? 0 : 1;
			temp3 = j + 3> ymx ? 0 : 1;

			t0 = temp0;//qwt
			t1 = temp1;
			t2 = temp2;
			t3 = temp3;

			//���ݶ�ֵ����ֵ
			temp0 *= psrcgray[j   *temp0 + i * devpar.ImgMakeborderWidth]>mThreshold ? psrcgray[j   *temp0 + i * devpar.ImgMakeborderWidth] : 0;
			temp1 *= psrcgray[(j + 1)*temp1 + i * devpar.ImgMakeborderWidth]>mThreshold ? psrcgray[(j + 1)*temp1 + i * devpar.ImgMakeborderWidth] : 0;
			temp2 *= psrcgray[(j + 2)*temp2 + i * devpar.ImgMakeborderWidth]>mThreshold ? psrcgray[(j + 2)*temp2 + i * devpar.ImgMakeborderWidth] : 0;
			temp3 *= psrcgray[(j + 3)*temp3 + i * devpar.ImgMakeborderWidth]>mThreshold ? psrcgray[(j + 3)*temp3 + i * devpar.ImgMakeborderWidth] : 0;

			t0 *= psrccounter[j   *t0 + i * devpar.ImgMakeborderWidth];
			t1 *= psrccounter[(j + 1)*t1 + i * devpar.ImgMakeborderWidth];
			t2 *= psrccounter[(j + 2)*t2 + i * devpar.ImgMakeborderWidth];
			t3 *= psrccounter[(j + 3)*t3 + i * devpar.ImgMakeborderWidth];


			count += temp0 > 0 ? 1 : 0; //�������
			count += temp1 > 0 ? 1 : 0;
			count += temp2 > 0 ? 1 : 0;
			count += temp3 > 0 ? 1 : 0;


			clengthCount += t0 + t1 + t2 + t3;//�ܳ�����
			sum_gray += temp0 + temp1 + temp2 + temp3;


			x_sum += i * temp0 + i * temp1 + i * temp2 + i * temp3;
			y_sum += j * temp0 + (j + 1)*temp1 + (j + 2)*temp2 + (j + 3)*temp3;
		}
	//ɸѡ����
	clength = clengthCount / 255;
	length[Id] = clength;
	area[Id] = count;
	xpos[Id] = x_sum / sum_gray;
	ypos[Id] = y_sum / sum_gray;
}

//-------------------------------------------------------����----------------------------------------//
void GPUDeviceCheck()
{
	hipError_t hipGetDeviceCount(int* count);
	hipGetDeviceCount(&gDeviceCount);
	for (int i = 0; i<gDeviceCount; i++)
	{
		hipDeviceProp_t DevProp;
		hipGetDeviceProperties(&DevProp, i);
		printf("Device %d has compute capability %d.%d \n", i, DevProp.major, DevProp.minor);
	}
}
// 8λ�Ҷ�BMP��ʽͼ���ȡ
bool RmwRead8BitBmpFile2Img(const char * filename, unsigned char*pImg, unsigned char*Binarization, int *width, int *height)
{
	FILE *binFile;
	BITMAPFILEHEADER fileHeader;//�ļ�ͷ
	BITMAPINFOHEADER bmpHeader;//��Ϣͷ
	BOOL isRead = TRUE;
	int ImgDeep;
	int linenum, ex; // nenum:һ�����ص��ֽ���������������ֽ�

		//open file
		if ((binFile = fopen(filename, "rb")) == NULL) return NULL;

	//read struts
	if (fread((void *)&fileHeader, 1, sizeof(fileHeader), binFile) != sizeof(fileHeader)) isRead = FALSE;
	if (fread((void *)&bmpHeader, 1, sizeof(bmpHeader), binFile) != sizeof(bmpHeader)) isRead = FALSE;

	if (isRead == FALSE || fileHeader.bfOffBits<sizeof(fileHeader) + sizeof(bmpHeader)) {
		fclose(binFile);
		return NULL;
	}

	//read image info
	*width = bmpHeader.biWidth;
	*height = bmpHeader.biHeight;
	ImgDeep = bmpHeader.biBitCount / 8;//ÿ��������ռ�ֽ���Ŀ
	linenum = (*width * ImgDeep + 3) / 4 * 4;//����Ҫ��
	ex = linenum - *width * ImgDeep;   //ÿһ�е�����ֽ�

	fseek(binFile, fileHeader.bfOffBits, SEEK_SET);
	//��ȡ�Ҷ�ͼ
	if (ImgDeep == 1)
	{
		if (Binarization != NULL)
			for (int i = 0; i<*height; i++)
			{
				int r = fread(Binarization + (*height - i - 1)*(*width)*ImgDeep, sizeof(unsigned char), (*width)*ImgDeep, binFile);
				if (r != (*width)*ImgDeep)
				{
					delete Binarization;
					fclose(binFile);
					return NULL;
				}
				fseek(binFile, ex, SEEK_CUR);
			}
		fclose(binFile);
		return 1;
	}
	//��ȡλͼ
	else if (ImgDeep == 3)
	{
		//pImg = new uchar[(*width)*(*height)*ImgDeep];
		if (pImg != NULL)
		{
			for (int i = 0; i < *height; i++)
			{
				int r = fread(pImg + (*height - i - 1)*(*width)*ImgDeep, sizeof(unsigned char), (*width)*ImgDeep, binFile);//**
				if (r != (*width)*ImgDeep)//**
				{
					fclose(binFile);
					return NULL;
				}
				fseek(binFile, ex, SEEK_CUR);
			}
			fclose(binFile);
			//bmpת�Ҷ�
			if (Binarization != NULL)
			{
				for (int i = 0; i < *height; i++)
					for (int j = 0; j < *width; j++)
					{
						Binarization[j + i * (*width)] = pImg[j * ImgDeep + i * (*width) * ImgDeep] * 0.299 +
							pImg[j * ImgDeep + 1 + i * (*width) * ImgDeep] * 0.587 +
							pImg[j * ImgDeep + 2 + i * (*width) * ImgDeep] * 0.114;
					}
			}
			return 1;
		}
		else//
		{
			unsigned char *tempImg = new uchar[(*width)*(*height)*ImgDeep];
			if (tempImg != NULL)
			{
				for (int i = 0; i < *height; i++)
				{
					int r = fread(tempImg + (*height - i - 1)*(*width)*ImgDeep, sizeof(unsigned char), (*width)*ImgDeep, binFile);//**
					if (r != (*width)*ImgDeep)//**
					{
						delete[]tempImg;
						fclose(binFile);
						return NULL;
					}
					fseek(binFile, ex, SEEK_CUR);
				}
				fclose(binFile);
				//bmpת�Ҷ�
				if (Binarization != NULL)
				{
					for (int i = 0; i < *height; i++)
						for (int j = 0; j < *width; j++)
						{
							Binarization[j + i * (*width)] = tempImg[j * ImgDeep + i * (*width) * ImgDeep] * 0.299 +
								tempImg[j * ImgDeep + 1 + i * (*width) * ImgDeep] * 0.587 +
								tempImg[j * ImgDeep + 2 + i * (*width) * ImgDeep] * 0.114;
						}
				}
				delete[]tempImg;
				return 1;
			}
		}
	}
	else return 0;
}

//����������ȡ��λ��---��ȡ��λ�����ظ�
void GetImgBoxHost(const char *path)
{
	Parameter devpar;
	//��ʼ��ͼ����Ϣ����
	devpar.ImgHeight = gStructVarible.ImgHeight;
	devpar.ImgWidth = gStructVarible.ImgWidth;
	devpar.Threshold = gStructVarible.Threshold;
	devpar.LengthMin = gStructVarible.LengthMin;
	devpar.LengthMax = gStructVarible.LengthMax;
	devpar.AreaMin = gStructVarible.AreaMin;
	devpar.AreaMax = gStructVarible.AreaMax;
	devpar.PictureNum = gStructVarible.PictureNum;

	//��λ��������
	const cv::Point directions[8] = { { 0, 1 },{ 1,1 },{ 1, 0 },{ 1, -1 },{ 0, -1 },{ -1, -1 },{ -1, 0 },{ -1, 1 } };
	//��ʼ��CPU�˷�λ������
	if (gHostRecData.size() != 0)
		gHostRecData.clear();
	//��ʼ��  ��λ�и�������
	gImgXcenter = 0;
	gImgYcenter = 0;
	gXcenterOffset = 0;
	gYcenterOffset = 0;
	//ͼ��ռ����
	unsigned char *ImgHostdata = new unsigned char[devpar.ImgWidth* devpar.ImgHeight*devpar.PictureNum]; //qwt���������BUG
	unsigned char *m_ptr = new unsigned char[devpar.ImgWidth* devpar.ImgHeight*devpar.PictureNum];//��ֵ��ͼ
	unsigned char *n_ptr = new unsigned char[devpar.ImgWidth* devpar.ImgHeight*devpar.PictureNum];//����ͼ
	unsigned char *c_ptr = new unsigned char[devpar.ImgWidth* devpar.ImgHeight*devpar.PictureNum];//����ͼ	
	
	int Picoffset = devpar.ImgHeight * devpar.ImgWidth;//qwt//��ȡͼƬ
	for (int j = 0; j < devpar.PictureNum; j++)
	{
		RmwRead8BitBmpFile2Img(path,NULL, ImgHostdata + j*Picoffset, &devpar.ImgWidth, &devpar.ImgHeight);//qwt823
	}
	//��ֵ��
	for (int i = 0; i <devpar.ImgHeight*devpar.PictureNum; i++)
	{
		for (int j = 0; j < devpar.ImgWidth; j++)
		{
			m_ptr[j + i * devpar.ImgWidth] = ImgHostdata[j + i * devpar.ImgWidth] > devpar.Threshold ? 255 : 0;
			c_ptr[j + i * devpar.ImgWidth] = m_ptr[j + i * devpar.ImgWidth];
			n_ptr[j + i * devpar.ImgWidth] = m_ptr[j + i * devpar.ImgWidth];
		}

	}
	//����
	for (int i = 1; i<devpar.ImgHeight*devpar.PictureNum - 1; i++)
		for (int j = 1; j <devpar.ImgWidth - 1; j++)
		{
			if (m_ptr[j + i * devpar.ImgWidth] == 0)
			{
				if (m_ptr[j - 1 + (i - 1)*devpar.ImgWidth] != 0 || m_ptr[j + (i - 1)*devpar.ImgWidth] != 0 || m_ptr[j + 1 + (i - 1)*devpar.ImgWidth] != 0 ||
					m_ptr[j - 1 + i * devpar.ImgWidth] != 0 || m_ptr[j + 1 + i * devpar.ImgWidth] != 0 ||
					m_ptr[j - 1 + (i + 1)*devpar.ImgWidth] != 0 || m_ptr[j + (i + 1)*devpar.ImgWidth] != 0 || m_ptr[j + 1 + (i + 1)*devpar.ImgWidth] != 0)
				{
					n_ptr[j + i * devpar.ImgWidth] = 255;
					c_ptr[j + i * devpar.ImgWidth] = 255;
				}
			}
		}
	//��ʴ  c_ptr������
	for (int i = 1; i<devpar.ImgHeight*devpar.PictureNum - 1; i++)
		for (int j = 1; j < devpar.ImgWidth - 1; j++)
		{
			if (n_ptr[j + i * devpar.ImgWidth] != 0)
			{
				if (n_ptr[j + (i - 1)*devpar.ImgWidth] != 0 && n_ptr[j - 1 + i * devpar.ImgWidth] != 0 &&
					n_ptr[j + 1 + i * devpar.ImgWidth] != 0 && n_ptr[j + (i + 1)*devpar.ImgWidth] != 0)
				{
					c_ptr[j + i * devpar.ImgWidth] = 0;
				}
			}
		}
	//��λ��
	short xmax;
	short xmin;
	short ymax;
	short ymin;
	// ��Ե����  
	int i, j, counts = 0, curr_d = 0;//counts����ѭ������   curr_d�Ƿ������������ID
	short cLength;
	//��ȡ��λ����
	for (i = 1; i <devpar.ImgHeight*devpar.PictureNum - 1; i++)
		for (j = 1; j <devpar.ImgWidth - 1; j++)
		{
			// ��ʼ�㼰��ǰ��  
			cv::Point b_pt = cv::Point(i, j);
			cv::Point c_pt = cv::Point(i, j);
			// �����ǰ��Ϊǰ����  
			if (255 == c_ptr[j + i * devpar.ImgWidth])
			{
				cLength = 1;
				xmin = xmax = i;
				ymin = ymax = j;

				bool first_t = false;
				bool tra_flag = false;//���ñ�־λ
									  // ����  
				c_ptr[j + i * devpar.ImgWidth] = 0;    // �ù��ĵ�ֱ�Ӹ�����Ϊ0  

													   // ���и���  
				while (!tra_flag)
				{
					// ѭ���˴�  
					for (counts = 0; counts < 8; counts++)
					{
						// ��ֹ��������  
						if (curr_d >= 8)
						{
							curr_d -= 8;
						}
						if (curr_d < 0)
						{
							curr_d += 8;
						}
						if (counts == 6 && first_t)
						{
							first_t = true;
							continue;
						}

						// ���ٵĹ��̣�Ӧ���Ǹ������Ĺ��̣���Ҫ��ͣ�ĸ���������root��  
						c_pt = cv::Point(b_pt.x + directions[curr_d].x, b_pt.y + directions[curr_d].y);

						// �߽��ж�  
						if ((c_pt.x > 0) && (c_pt.x < devpar.ImgHeight*devpar.PictureNum - 1) &&
							(c_pt.y > 0) && (c_pt.y < devpar.ImgWidth - 1))
						{
							// ������ڱ�Ե  
							if (255 == c_ptr[c_pt.x*devpar.ImgWidth + c_pt.y])
							{
								//���°�Χ��
								xmax = xmax > c_pt.x ? xmax : c_pt.x;
								ymax = ymax > c_pt.y ? ymax : c_pt.y;
								xmin = xmin < c_pt.x ? xmin : c_pt.x;
								ymin = ymin < c_pt.y ? ymin : c_pt.y;
								curr_d -= 2;   //���µ�ǰ����  
								c_ptr[c_pt.x*devpar.ImgWidth + c_pt.y] = 0;
								// ����b_pt:���ٵ�root��  
								b_pt.x = c_pt.x;
								b_pt.y = c_pt.y;
								cLength++;
								break;   // ����forѭ��  
							}
						}
						curr_d++;
					}   // end for  
						// ���ٵ���ֹ���������8���򶼲����ڱ�Ե  
					if (8 == counts)
					{
						// ����  
						curr_d = 0;
						tra_flag = true;
						if (cLength<devpar.LengthMax && (cLength >devpar.LengthMin))
						{
							RecData tempRecData;
							if (xmin - 3 < 0)
								tempRecData.RecXmin = 0;
							else
								tempRecData.RecXmin = xmin - 3;
							if (ymin - 3 < 0)
								tempRecData.RecYmin = 0;
							else
								tempRecData.RecYmin = ymin - 3;
							if (xmax + 3 >devpar.ImgHeight*devpar.PictureNum-1)
								tempRecData.RecXmax = devpar.ImgHeight*devpar.PictureNum - 1;
							else
								tempRecData.RecXmax = xmax + 3;
							if (ymax + 3 > devpar.ImgWidth)
								tempRecData.RecYmax = devpar.ImgWidth - 1;
							else
								tempRecData.RecYmax = ymax + 3;
							if (abs((tempRecData.RecYmax - tempRecData.RecYmin) - (tempRecData.RecXmax - tempRecData.RecXmin))<20)
								gHostRecData.push_back(tempRecData);
						}
						break;
					}
				}  // end if  
			}  // end while  
		}
	//��ʼ�� ��λ�и�������
	if (gHostRecData.size() > 0)
	{
		//��ȡͼ���־�������ۼӺ�
		for (int k = 0; k < gHostRecData.size(); k++)
		{
			//�������κлҶ����ļ���
			int GraySum = 0;
			int xGraySum = 0;
			int yGraySum = 0;
			for (int i = gHostRecData[k].RecXmin + 3; i <= gHostRecData[k].RecXmax - 3; i++)
				for (int j = gHostRecData[k].RecYmin + 3; j <= gHostRecData[k].RecYmax - 3; j++)
				{
					if (ImgHostdata[j + i * devpar.ImgWidth] >(devpar.Threshold))
					{
						xGraySum += i * ImgHostdata[j + i * devpar.ImgWidth];
						yGraySum += j * ImgHostdata[j + i * devpar.ImgWidth];
						GraySum += ImgHostdata[j + i * devpar.ImgWidth];
					}
				}
			gImgXcenter += xGraySum / GraySum;
			gImgYcenter += yGraySum / GraySum;
		}
		//�����Ȩ�Ҷ�����
		gImgXcenter /= gHostRecData.size();
		gImgYcenter /= gHostRecData.size();
		//����ƫ������ʼ��
		gXcenterOffset = 0;
		gYcenterOffset = 0;
		gRectRealNum = gHostRecData.size();//��ȡ��λ��ʵ������ֵ
		//������λ�����������ú����߳�����
		int rRecNum = (gHostRecData.size() + 127) / 128 * 128;
		gHostRecData.resize(rRecNum, RecData{ 0,0,0,0 });
	}
	//�ͷ��ڴ�
	delete[]ImgHostdata;
	delete[]m_ptr;
	delete[]n_ptr;
	delete[]c_ptr;
}

//qwt7.26
class R : public Runnable
{
public:
	Parameter Devpar;//��������
	~R()
	{
	}
	void Run()
	{
		//������
		hipError_t  err;
		int img_index = 0;
		int Width;
		int Height;
		char strFilename[100];                                          //��1������һ���ַ����鱣��----ͼƬ�Ķ�ȡ·�� 
		char saveFilename[100];                                         //��1������һ���ַ����鱣��----ͼƬ�Ĵ洢·��
		char* path = "E:\\project\\Simulation\\output";


		//��������
		Devpar.ImgMakeborderWidth = (Devpar.ImgWidth + 127) / 128 * 128;//����Ŀ�ȼ���
		Devpar.RowThreadNum = Devpar.ImgHeight*Devpar.PictureNum / 8;
		Devpar.ColThreadNum = (Devpar.ImgWidth / 8 + 127) / 128 * 128;

		dim3 mGrid1(Devpar.ImgMakeborderWidth / 128, Devpar.ImgHeight*Devpar.PictureNum, 1);
		dim3 mGrid2(Devpar.ColThreadNum / 128, Devpar.RowThreadNum, 1);

		//����GPU�豸��
		/*������*/
		//����
		short *gpHostLength[CUDAStreams];
		short *gpHostArea[CUDAStreams];
		short *gpHostXpos[CUDAStreams];
		short *gpHostYpos[CUDAStreams];
		short *gpHostIndex[CUDAStreams];
		/*�豸��*/
		short *  gpDevRecXLeft[CUDAStreams];
		short *  gpDevRecYLeft[CUDAStreams];
		short *  gpDevRecXRight[CUDAStreams];
		short *  gpDevRecYRight[CUDAStreams];
		//���
		short  *gpDevLength[CUDAStreams];
		short  *gpDevArea[CUDAStreams];
		short  *gpDevXpos[CUDAStreams];
		short  *gpDevYpos[CUDAStreams];
		short  *gpDevIndex[CUDAStreams];

		//�������ʱ�����ռ䣬�����з�λ�С����������GPU���ڴ��GPU�Դ�
		for (int i = 0; i < CUDAStreams; i++)
		{
			checkCudaErrors(hipHostAlloc((void**)&gpHostLength[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short), hipHostMallocDefault));//����ܳ�
			checkCudaErrors(hipHostAlloc((void**)&gpHostArea[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short), hipHostMallocDefault));//���
			checkCudaErrors(hipHostAlloc((void**)&gpHostXpos[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short), hipHostMallocDefault));//��������x
			checkCudaErrors(hipHostAlloc((void**)&gpHostYpos[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short), hipHostMallocDefault));//��������y
			checkCudaErrors(hipHostAlloc((void**)&gpHostIndex[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short), hipHostMallocDefault));//����������
			checkCudaErrors(hipMalloc((void**)&gpDevRecXLeft[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//��λ�� xmin
			checkCudaErrors(hipMalloc((void**)&gpDevRecYLeft[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//	    ymin
			checkCudaErrors(hipMalloc((void**)&gpDevRecXRight[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//		xmax
			checkCudaErrors(hipMalloc((void**)&gpDevRecYRight[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//		ymax
			checkCudaErrors(hipMalloc((void**)&gpDevLength[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//�豸�����	�ܳ�
			checkCudaErrors(hipMalloc((void**)&gpDevArea[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//				���
			checkCudaErrors(hipMalloc((void**)&gpDevXpos[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//				xpos
			checkCudaErrors(hipMalloc((void**)&gpDevYpos[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//				ypos
			checkCudaErrors(hipMalloc((void**)&gpDevIndex[i], Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short)));//				������
		}

		while ((img_index + CUDAStreams) <= gHostPathImgNumber)
		{
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(rDev_in[i], gHostImage[img_index + i], sizeof(uchar)* Devpar.ImgHeight *Devpar.ImgWidth*Devpar.PictureNum, hipMemcpyHostToDevice, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				CopyMakeBorder << <mGrid1, 128, 0, rcS[i] >> > (rDev_in[i], rDev_padding[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				Binarization << <mGrid1, 128, 0, rcS[i] >> > (rDev_padding[i], rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//�߽���ȡ
				Dilation << <mGrid1, 128, 0, rcS[i] >> >(rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
			hipMemcpyAsync(rgpu_2val[i], rgpu_counter[i], sizeof(uchar)* Devpar.ImgHeight *Devpar.ImgMakeborderWidth*Devpar.PictureNum, hipMemcpyDeviceToDevice, rcS[i]);
			}
		
			for (int i = 0; i < CUDAStreams; i++)
			{
				Erosion << <mGrid1, 128, 0, rcS[i] >> > (rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ�����ͱ�Ե��
				GetCounter << <mGrid2, 128, 0, rcS[i] >> > (rgpu_counter[i], gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i], Devpar);//��ȡ�����ĺ���
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ���������//��ȡ������Ϣ�˺���
				GetInfo << <mGrid2, 128, 0, rcS[i] >> > (rDev_padding[i],  gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i], gpDevXpos[i], gpDevYpos[i], gpDevArea[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ɸѡ��ȡ������������ķ��ظ���Ϣ
				GetTrueInfo << <mGrid2, 128, 0, rcS[i] >> > (gpDevXpos[i], gpDevYpos[i], gpDevIndex[i], gpDevArea[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				err=hipMemcpyAsync(gpHostLength[i], gpDevLength[i], sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			printf("%s\n", hipGetErrorString(err));
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostArea[i], gpDevArea[i], sizeof(short)*	Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostXpos[i], gpDevXpos[i], sizeof(short)*	Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostYpos[i], gpDevYpos[i], sizeof(short)*	Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostIndex[i], gpDevIndex[i], sizeof(short)*	Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipStreamSynchronize(rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ɸѡ��ӡ��ȡ������
				vector<CircleInfo>myInfo;
				for (int j = 0; j < Devpar.ColThreadNum * Devpar.RowThreadNum; j++)
				{
					if (gpHostIndex[i][j] != 0)
					{
						CircleInfo temp;
						temp.index = (short)j;
						temp.length = gpHostLength[i][j];
						temp.area = gpHostArea[i][j];
						temp.xpos = gpHostXpos[i][j];
						temp.ypos = gpHostYpos[i][j];
						myInfo.push_back(temp);
					}
				}
				SignPoint.PointNumbers = myInfo.size();
				//�����־������
				if (myInfo.size() > 0) 
				{
					FILE* fp;
					sprintf_s(strFilename, "%s\\%d.bin", path, img_index + i + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ�
					fp = fopen(strFilename, "wb");
					fwrite(&myInfo[0], sizeof(CircleInfo)*myInfo.size(), 1, fp);
					fclose(fp);
				}
			}
			img_index += gHostImgblock;
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipHostFree(gpHostLength[i]);
			hipHostFree(gpHostArea[i]);
			hipHostFree(gpHostXpos[i]);
			hipHostFree(gpHostYpos[i]);
			hipHostFree(gpHostIndex[i]);
			//�豸���ڴ�
			hipFree(gpDevRecXLeft[i]);
			hipFree(gpDevRecYLeft[i]);
			hipFree(gpDevRecXRight[i]);
			hipFree(gpDevRecYRight[i]);
			hipFree(gpDevLength[i]);
			hipFree(gpDevArea[i]);
			hipFree(gpDevXpos[i]);
			hipFree(gpDevYpos[i]);
			hipFree(gpDevIndex[i]);
		}
	}
};

//����ģʽ����R
class RecR : public Runnable
{
public:
	Parameter Devpar;//��������	
	~RecR()
	{
	}
	void Run()
	{
		hipError_t err;
		int xCenterSum = 0;//���·�λ���������ݣ�
		int yCenterSum = 0;//���·�λ����������
		int img_index = 0;
		char strFilename[100];                                          //��1������һ���ַ����鱣��----ͼƬ�Ķ�ȡ·�� 
		char saveFilename[100];                                         //��1������һ���ַ����鱣��----ͼƬ�Ĵ洢·��
		char* path = "E:\\project\\Simulation\\output";
		//��������
		Devpar.ImgMakeborderWidth = (Devpar.ImgWidth + 127) / 128 * 128;//����Ŀ�ȼ���
		int mRecCount = gHostRecData.size();//��ȡ��λ������
		//�˺�������GRid���䣻
		int Gridsize = mRecCount / 128;
		if (Gridsize == 0)//qwt823
			Gridsize = 1;
		dim3 mGrid1(Devpar.ImgMakeborderWidth / 128, Devpar.ImgHeight*Devpar.PictureNum, 1);
		dim3 mGrid2(Gridsize, 1, 1);

		/*������*/
		//����
		short *gpHostLength[CUDAStreams];
		short *gpHostArea[CUDAStreams];
		short *gpHostXpos[CUDAStreams];
		short *gpHostYpos[CUDAStreams];
		//���
		short  *gpDevLength[CUDAStreams];
		short  *gpDevArea[CUDAStreams];
		short  *gpDevXpos[CUDAStreams];
		short  *gpDevYpos[CUDAStreams];
		//������λ������
		RecData *gpRDevRecData[CUDAStreams];//qwt821
		if (gHostRecData.size() > 0) 
		{
			for (int i = 0; i < CUDAStreams; i++)
			{
				checkCudaErrors(hipMalloc((void**)&gpRDevRecData[i], mRecCount * sizeof(RecData)));//
				hipMemcpy(gpRDevRecData[i], &gHostRecData[0], mRecCount * sizeof(RecData), hipMemcpyHostToDevice);
			}
		}
		//�洢�ռ����
		for (int i = 0; i < CUDAStreams; i++)
		{
			checkCudaErrors(hipHostAlloc((void**)&gpHostLength[i], mRecCount * sizeof(short), hipHostMallocDefault));//����ܳ�
			checkCudaErrors(hipHostAlloc((void**)&gpHostArea[i], mRecCount * sizeof(short), hipHostMallocDefault));//���
			checkCudaErrors(hipHostAlloc((void**)&gpHostXpos[i], mRecCount * sizeof(short), hipHostMallocDefault));//��������x
			checkCudaErrors(hipHostAlloc((void**)&gpHostYpos[i], mRecCount * sizeof(short), hipHostMallocDefault));//��������y
			checkCudaErrors(hipMalloc((void**)&gpDevLength[i], mRecCount * sizeof(short)));//�豸�����	�ܳ�
			checkCudaErrors(hipMalloc((void**)&gpDevArea[i], mRecCount * sizeof(short)));//				���
			checkCudaErrors(hipMalloc((void**)&gpDevXpos[i], mRecCount * sizeof(short)));//				xpos
			checkCudaErrors(hipMalloc((void**)&gpDevYpos[i], mRecCount * sizeof(short)));//				ypos
		}
		while ((img_index + CUDAStreams ) <= gHostPathImgNumber)
		{
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(rDev_in[i], gHostImage[img_index + i ], sizeof(unsigned char)* Devpar.ImgHeight * Devpar.ImgWidth*Devpar.PictureNum, hipMemcpyHostToDevice, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				CopyMakeBorder << <mGrid1, 128, 0, rcS[i] >> > (rDev_in[i], rDev_padding[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				Binarization << <mGrid1, 128, 0, rcS[i] >> > (rDev_padding[i],  rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//�߽���ȡ
				Dilation << <mGrid1, 128, 0, rcS[i] >> >(rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(rgpu_2val[i], rgpu_counter[i], sizeof(unsigned char)* Devpar.ImgHeight *Devpar.ImgMakeborderWidth*Devpar.PictureNum, hipMemcpyDeviceToDevice, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				Erosion << <mGrid1, 128, 0, rcS[i] >> > (rgpu_2val[i], rgpu_counter[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ͬ���еĺ˺�����ͬһGPU����ʱ���Ƿ��Ӱ��˺���������qwt
				GetRecInfo << <mGrid2, 128, 0, rcS[i] >> >(gpRDevRecData[i], rDev_padding[i], rgpu_counter[i],
					gpDevLength[i], gpDevArea[i], gpDevXpos[i], gpDevYpos[i], Devpar);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostLength[i], gpDevLength[i], sizeof(short)*   mRecCount, hipMemcpyDeviceToHost, rcS[i]);
			}
			//printf(" %s\n", hipGetErrorString(err));//������
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostArea[i], gpDevArea[i], sizeof(short)*   mRecCount, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostXpos[i], gpDevXpos[i], sizeof(short)*  mRecCount, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				err=hipMemcpyAsync(gpHostYpos[i], gpDevYpos[i], sizeof(short)*  mRecCount, hipMemcpyDeviceToHost, rcS[i]);
			}
		//	printf(" %s\n", hipGetErrorString(err));//������
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipStreamSynchronize(rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ɸѡ��ӡ��ȡ������
				vector<CircleInfo>myInfo;
				xCenterSum = 0;
				yCenterSum = 0;
				for (int j = 0; j < mRecCount; j++)
				{
					if (gpHostXpos[i][j] > 0) 
					{
						CircleInfo temp;
						temp.index = j;
						temp.length = gpHostLength[i][j];
						temp.area = gpHostArea[i][j];
						temp.xpos = gpHostXpos[i][j];
						temp.ypos = gpHostYpos[i][j];
						xCenterSum += gpHostXpos[i][j];//�����ۼӣ�x
						yCenterSum += gpHostYpos[i][j];//�����ۼӣ�y
						myInfo.push_back(temp);
					}
				}
				//��Χ����Ϣ����
				if (myInfo.size() > 0) 
				{
					gXcenterOffset += (gImgXcenter - xCenterSum / myInfo.size());
					gYcenterOffset += (gImgYcenter - yCenterSum / myInfo.size());
					gImgXcenter = xCenterSum / myInfo.size();
					gImgYcenter = yCenterSum / myInfo.size();
				}
				SignPoint.PointNumbers = myInfo.size();
				//�����־������
				if (myInfo.size() > 0)
				{
					FILE* fp;
					sprintf_s(strFilename, "%s\\%d.bin", path, img_index + i + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ�
					fp = fopen(strFilename, "wb");
					fwrite(&myInfo[0], sizeof(CircleInfo)*myInfo.size(), 1, fp);
					fclose(fp);
				}
			}
			img_index += gHostImgblock;
			//���°�Χ��
			if (img_index % 100 == 0)
			{
				for (int i = 0; i < mRecCount; i++)
				{
					if (gHostRecData[i].RecXmin != 0 && gHostRecData[i].RecYmin != 0)
					{
						gHostRecData[i].RecXmin += gXcenterOffset;//CPU�˷�λ��
						gHostRecData[i].RecXmax += gXcenterOffset;
						gHostRecData[i].RecYmin += gYcenterOffset;
						gHostRecData[i].RecYmax += gYcenterOffset;
					}
					//�߽��ж�
					if (gHostRecData[i].RecXmin<0 || gHostRecData[i].RecXmin>Devpar.ImgHeight* Devpar.PictureNum||
						gHostRecData[i].RecXmax<0 || gHostRecData[i].RecXmax>Devpar.ImgHeight* Devpar.PictureNum ||
						gHostRecData[i].RecYmin<0 || gHostRecData[i].RecYmin>Devpar.ImgMakeborderWidth ||
						gHostRecData[i].RecYmax<0 || gHostRecData[i].RecYmax>Devpar.ImgMakeborderWidth)
					{
						gHostRecData[i].RecXmin = 0;
						gHostRecData[i].RecXmax = 0;
						gHostRecData[i].RecYmin = 0;
						gHostRecData[i].RecYmax = 0;
					}
				}
				gXcenterOffset = 0;
				gYcenterOffset = 0;
				for (int i = 0; i < CUDAStreams; i++)
				{
					hipMemcpy(gpRDevRecData[i], &gHostRecData[0], mRecCount * sizeof(RecData), hipMemcpyHostToDevice);
				}
			}
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipHostFree(gpHostLength[i]);
			hipHostFree(gpHostArea[i]);
			hipHostFree(gpHostXpos[i]);
			hipHostFree(gpHostYpos[i]);
			//�豸���ڴ�
			hipFree(gpDevLength[i]);
			hipFree(gpDevArea[i]);
			hipFree(gpDevXpos[i]);
			hipFree(gpDevYpos[i]);
			hipFree(gpRDevRecData[i]);
		}
	}
};
//����ģʽ����S

//����ԭͼ�������
IMGSIMULATION_API bool SimulationImageTest(const char *path, Infomation *Info) 
{
	hipError_t  err;
	int mWidth, mHeight;
	gHostPathImgNumber = 20;//����ͼƬ��������
	for (int i = 0; i < gHostPathImgNumber; i++)
	{
		err = hipHostAlloc((void**)&gHostImage[i], gStructVarible.ImgHeight * gStructVarible.ImgWidth *gStructVarible.PictureNum* sizeof(unsigned char), hipHostMallocDefault);
	}
	int Picoffset = gStructVarible.ImgHeight * gStructVarible.ImgWidth;//����ͼƬ��ַƫ����
	for (int i = 0; i < gHostPathImgNumber; i++)
	{
		for (int j = 0; j < gStructVarible.PictureNum; j++)
		{
			RmwRead8BitBmpFile2Img(path, NULL, gHostImage[i]+j*Picoffset, &mWidth, &mHeight);
		}
	}
	//����ͼƬ�Ƿ��ȡ�ɹ�------------------------------------------------------------------------------------------------------------------
	//cv::Mat img(gStructVarible.ImgHeight*gStructVarible.PictureNum, gStructVarible.ImgWidth, CV_8UC1);
	//for (int i = 0; i < gStructVarible.ImgHeight*gStructVarible.PictureNum; i++)
	//{
	//	uchar* data = img.ptr<uchar>(i);  //��ȡ��i�е��׵�ַ��
	//	for (int j = 0; j < gStructVarible.ImgWidth; j++)   //��ѭ��
	//	{
	//		data[j] = gHostImage[10][j + i *   gStructVarible.ImgWidth];
	//	}
	//}
	//imwrite("pic.bmp", img);
	//-------------------------------------------------------------------------------------------------------------------------------------
	if(gStructVarible.RecModelFlag == true)
		 GetImgBoxHost(path);//��ȡ��Χ��
	cout << gHostRecData.size();
	/****  ��������****/
	CThreadPoolExecutor * pExecutor = new CThreadPoolExecutor();
	pExecutor->Init(1, ExtractPointThreads, 1);
	R r;
	RecR recr;

	if (gStructVarible.RecModelFlag == false)//ȫͼģʽ
	{
		//�ṹ�帳ֵ
		r.Devpar.ImgHeight = gStructVarible.ImgHeight;
		r.Devpar.ImgWidth = gStructVarible.ImgWidth;
		r.Devpar.Threshold = gStructVarible.Threshold;
		r.Devpar.LengthMin = gStructVarible.LengthMin;
		r.Devpar.LengthMax = gStructVarible.LengthMax;
		r.Devpar.AreaMin = gStructVarible.AreaMin;
		r.Devpar.AreaMax = gStructVarible.AreaMax;
		r.Devpar.PictureNum = gStructVarible.PictureNum;
		pExecutor->Execute(&r, 0x01);

		pExecutor->Terminate();
		delete pExecutor;
		
	}
	else //����ģʽ
	{
		//�ṹ�帳ֵ
		recr.Devpar.ImgHeight = gStructVarible.ImgHeight;
		recr.Devpar.ImgWidth = gStructVarible.ImgWidth;
		recr.Devpar.Threshold = gStructVarible.Threshold;
		recr.Devpar.LengthMin = gStructVarible.LengthMin;
		recr.Devpar.LengthMax = gStructVarible.LengthMax;
		recr.Devpar.AreaMin = gStructVarible.AreaMin;
		recr.Devpar.AreaMax = gStructVarible.AreaMax;
		recr.Devpar.PictureNum = gStructVarible.PictureNum;
		
		pExecutor->Execute(&recr, 0x01);
		pExecutor->Terminate();
		delete pExecutor;
		
	}
	for (int i = 0; i < gHostPathImgNumber; i++)
	{
		err = hipHostFree(gHostImage[i]);
		if (gStructVarible.ImgBitDeep == 24)
		{
			delete(gHostColorImage[i]);
		}
		if (err != hipSuccess)
		{
			return false;
		}
	}
	if (gStructVarible.RecModelFlag == false)//ȫͼģʽ
	{
		return false;
	}
	else if(gStructVarible.RecModelFlag == true)//����ģʽ 
	{
		return true;
	}
}


//ȫ���ڴ�����
IMGSIMULATION_API void Memory_application(Parameter Devpar)
{
	int paddingWidth = (Devpar.ImgWidth  + 127) / 128 * 128;  //qwt7.26
	hipSetDevice(0);
	rcS = (hipStream_t *)malloc(CUDAStreams * sizeof(hipStream_t));
	for (int i = 0; i < CUDAStreams; i++)
	{
		hipSetDevice(0);
		hipStreamCreate(&(rcS[i]));
		hipMalloc((void**)&rDev_in[i],		 Devpar.ImgHeight *Devpar.ImgWidth  *Devpar.PictureNum* sizeof(unsigned char));
		hipMalloc((void**)&rDev_padding[i], Devpar.ImgHeight *paddingWidth		*Devpar.PictureNum* sizeof(unsigned char));  //qwt7.26
		hipMalloc((void**)&rgpu_2val[i],	 Devpar.ImgHeight *paddingWidth		*Devpar.PictureNum* sizeof(unsigned char));
		hipMalloc((void**)&rgpu_counter[i], Devpar.ImgHeight *paddingWidth		*Devpar.PictureNum* sizeof(unsigned char));
	}
}
//ȫ���ڴ��ͷ�
IMGSIMULATION_API void Memory_release()
{
	for (int i = 0; i<CUDAStreams; i++)
	{
		hipSetDevice(0);
		//hipHostFree(rhost_in[i]);
		hipFree(rDev_in[i]);
		hipFree(rDev_padding[i]); //qwt7.26
		hipFree(rgpu_2val[i]);
		hipFree(rgpu_counter[i]);
		checkCudaErrors(hipStreamDestroy(rcS[i]));
	}
}

int main()
{
	//��������
	gStructVarible.ImgReadPath = "E:\\project\\Simulation\\data_gray_img\\3.bmp";
	//gStructVarible.ImgSavePath = "E:\\project\\Simulation\\pic_output";
	gStructVarible.DataReadPath = "E:\\project\\Simulation\\output";
	gStructVarible.ImgHeight =5120;
	gStructVarible.ImgWidth = 5120;
	gStructVarible.Threshold =60;
	gStructVarible.LengthMin = 100;
	gStructVarible.LengthMax = 250;
	gStructVarible.AreaMin = 1;
	gStructVarible.AreaMax = 99999;
	gStructVarible.ImgBitDeep = 8;
	gStructVarible.PictureNum = 2;
	gStructVarible.RecModelFlag = false;
	
	////�����ڴ�ռ�
	Memory_application(gStructVarible);
	Infomation *result = new Infomation;
	SimulationImageTest(gStructVarible.ImgReadPath, result);
	delete result;
	//����
	char *strfilename = "E:\\project\\Simulation\\output\\1.bin";
	FILE *fr;
	fr = fopen(strfilename, "rb");
	if (fr == NULL)
	{
		cout << "FILE fail open" << endl;
		return 0;
	}
	fseek(fr, 0, SEEK_END);
	long lSize = ftell(fr);
	rewind(fr);
	int num9 = lSize / sizeof(CircleInfo);
	CircleInfo *RInfo = (CircleInfo*)malloc(sizeof(CircleInfo)*num9);
	fread(RInfo, sizeof(CircleInfo), num9, fr);
	fclose(fr);
	//����ԭ��ͼ
	cv::Mat img = cv::imread(gStructVarible.ImgReadPath, cv::IMREAD_COLOR);
	cv::Vec3b pflag(0, 0, 255);
	for (int i = 0; i < num9/gStructVarible.PictureNum; i++)
	{

		img.at<cv::Vec3b>(RInfo[i].xpos%gStructVarible.ImgHeight, RInfo[i].ypos%gStructVarible.ImgWidth) = pflag;
	}

	//���ư�Χ��
	/*for (int i = 0; i < gRectRealNum; i++)
	{
		cv::Point  Rmin(gHostRecData[i].RecYmin + 2, gHostRecData[i].RecXmin + 2);
		cv::Point  Rmax(gHostRecData[i].RecYmax - 2, gHostRecData[i].RecXmax - 2);
		rectangle(img, Rmin, Rmax, cv::Scalar(0, 0, 255));
	}*/
	return 0;
}

/*ͼ������������*/
//int main() 
//{
//	Mat img = imread("E:\\project\\Simulation\\data_gray_img\\4M.bmp",0);
//	Mat img1(2048, 2000, CV_8UC1);
//	for (int i = 0; i < img1.rows; i++)
//	{
//		uchar* data = img.ptr<uchar>(i);  //��ȡ��i�е��׵�ַ��
//		uchar* data1 = img1.ptr<uchar>(i);
//		for (int j = 0; j < img1.cols; j++)   //��ѭ��
//			{
//			if(i<(img1.rows-50)&&j<(img1.cols-50))
//				data1[j] = data[j];
//			else data1[j] = 0;
//		}
//		}
//	imwrite("E:\\project\\Simulation\\data_gray_img\\12.bmp", img1);
//	return 0;
//
//}



/*������ͨģʽ*/
//int main() 
//{
//	hipError_t err;
//	char *path = "E:\\project\\Simulation\\data_gray_img\\2.bmp";
//	const char * strfilename = "Sig1.bin";
//	Parameter Devpar;
//	Devpar.ImgHeight = 2400;
//	Devpar.ImgWidth = 1600;
//	Devpar.Threshold = 128;
//	Devpar.LengthMin = 30;
//	Devpar.LengthMax = 250;
//	Devpar.AreaMin = 1;
//	Devpar.AreaMax = 99999;
//	Devpar.ImgMakeborderWidth = (Devpar.ImgWidth + 127) / 128 * 128;
//	Devpar.ColThreadNum = (Devpar.ImgMakeborderWidth / 8 + 127) / 128 * 128;
//	Devpar.RowThreadNum = Devpar.ImgHeight / 8;
//	Devpar.PictureNum = 1;
//	// �߳����ö���
//	dim3 mGrid1(Devpar.ImgMakeborderWidth / 128, Devpar.ImgHeight, 1);
//	dim3 mGrid2(Devpar.ColThreadNum / 128, Devpar.RowThreadNum, 1);
//	unsigned char *tHostImage;
//	hipHostAlloc((void**)&tHostImage, Devpar.ImgHeight *  Devpar.ImgWidth * sizeof(unsigned char), hipHostMallocDefault);
//	RmwRead8BitBmpFile2Img(path,NULL,tHostImage, &Devpar.ImgWidth, &Devpar.ImgHeight);
//	
//	//���Զ���ͼƬ�Ƿ�ɹ�------------------------------------------------------------------------------------------------------------
//	cv::Mat img1(Devpar.ImgHeight, Devpar.ImgWidth, CV_8UC1);
//	for (int i = 0; i < Devpar.ImgHeight; i++)
//	{
//		uchar* data = img1.ptr<uchar>(i);  //��ȡ��i�е��׵�ַ��
//		for (int j = 0; j < Devpar.ImgWidth; j++)   //��ѭ��
//		{
//			data[j] = tHostImage[j + i * Devpar.ImgWidth];
//		}
//	}
//	//-------------------------------------------------------------------------------------------------------------------------------
//	unsigned char * tDevImage;
//	unsigned char * tDevpad;
//	unsigned char * tDev2val;
//	unsigned char * tDevcounter;
//	hipMalloc((void**)&tDevImage, sizeof(unsigned char)* Devpar.ImgWidth* Devpar.ImgHeight);
//	hipMalloc((void**)&tDevpad, sizeof(unsigned char)* Devpar.ImgMakeborderWidth* Devpar.ImgHeight);
//	hipMalloc((void**)&tDev2val, sizeof(unsigned char)* Devpar.ImgMakeborderWidth* Devpar.ImgHeight);
//	hipMalloc((void**)&tDevcounter, sizeof(unsigned char)* Devpar.ImgMakeborderWidth* Devpar.ImgHeight);
//	//�豸���Դ�����
//	short *  tDevRecXLeft;
//	short *  tDevRecYLeft;
//	short *  tDevRecXRight;
//	short *  tDevRecYRight;
//	short  *tDevLength;
//	short  *tDevArea;
//	short  *tDevXpos;
//	short  *tDevYpos;
//	short  *tDevIndex;
//	hipMalloc((void**)&tDevRecXLeft, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//��λ�� xmin
//	hipMalloc((void**)&tDevRecYLeft, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//	    ymin
//	hipMalloc((void**)&tDevRecXRight, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//		xmax
//	hipMalloc((void**)&tDevRecYRight, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//		ymax
//	hipMalloc((void**)&tDevLength, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//�豸�����	�ܳ�
//	hipMalloc((void**)&tDevArea, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//				���
//	hipMalloc((void**)&tDevXpos, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//				xpos
//	hipMalloc((void**)&tDevYpos, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//				ypos
//	hipMalloc((void**)&tDevIndex, Devpar.ColThreadNum*Devpar.RowThreadNum * sizeof(short));//				������
//																							//����ռ�����
//	short *  tHostRecXLeft = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostRecYLeft = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostRecXRight = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostRecYRight = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostLength = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostArea = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostXpos = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostYpos = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	short *  tHostIndex = new short[Devpar.ColThreadNum*Devpar.RowThreadNum];
//	//�˺���ִ��
//	hipMemcpy(tDevImage, tHostImage, sizeof(unsigned char)* Devpar.ImgHeight *Devpar.ImgWidth, hipMemcpyHostToDevice);
//	//ִ�лҶȻ�����ֵ���˺�������
//	CopyMakeBorder << <mGrid1, 128 >> > (tDevImage, tDevpad, Devpar);
//	//ִ�лҶȻ�����ֵ���˺�������
//	Binarization << <mGrid1, 128 >> > (tDevpad, tDev2val, tDevcounter, Devpar);
//	//�߽���ȡ
//	Dilation << <mGrid1, 128 >> > (tDev2val, tDevcounter, Devpar);
//	hipMemcpy(tDev2val, tDevcounter, sizeof(unsigned char)* Devpar.ImgHeight *Devpar.ImgMakeborderWidth, hipMemcpyDeviceToDevice);
//	Erosion << <mGrid1, 128 >> > (tDev2val, tDevcounter, Devpar);
//	//��ȡ�ܳ��Ͱ�Χ��
//	GetCounter << <mGrid2, 128 >> > (tDevcounter, tDevLength, tDevRecXLeft, tDevRecYLeft, tDevRecXRight, tDevRecYRight, Devpar);//��ȡ�����ĺ���																													//����ͼ��Ԥ�����Ƿ�ɹ�
//	GetInfo << <mGrid2, 128 >> > (tDevpad, tDevLength, tDevRecXLeft, tDevRecYLeft, tDevRecXRight, tDevRecYRight, tDevXpos, tDevYpos, tDevArea, Devpar);
//	GetTrueInfo << <mGrid2, 128 >> > (tDevXpos, tDevYpos, tDevIndex, tDevArea, Devpar);
//	//����������
//	hipMemcpy(tHostLength, tDevLength, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostArea, tDevArea, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostXpos, tDevXpos, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostYpos, tDevYpos, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostIndex, tDevIndex, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostRecXLeft, tDevRecXLeft, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostRecYLeft, tDevRecYLeft, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	hipMemcpy(tHostRecXRight, tDevRecXRight, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	err = hipMemcpy(tHostRecYRight, tDevRecYRight, sizeof(short)* Devpar.ColThreadNum * Devpar.RowThreadNum, hipMemcpyDeviceToHost);
//	printf("%s", hipGetErrorString(err));
//	//�������ͼ-----------------------------------------------------------------------------------------------------------------------
//	uchar* Src_counter = new uchar[Devpar.ImgHeight*Devpar.ImgMakeborderWidth];
//	cv::Mat img_counter(Devpar.ImgHeight, Devpar.ImgMakeborderWidth, CV_8UC1);
//	err = hipMemcpy(Src_counter, tDevcounter, sizeof(unsigned char)*Devpar.ImgHeight * Devpar.ImgMakeborderWidth, hipMemcpyDeviceToHost);
//	printf("%s", hipGetErrorString(err));
//	for (int i = 0; i < Devpar.ImgHeight; i++)
//	{
//		uchar* data = img_counter.ptr<uchar>(i);  //��ȡ��i�е��׵�ַ��
//		for (int j = 0; j < Devpar.ImgMakeborderWidth; j++)   //��ѭ��
//		{
//			data[j] = Src_counter[j + Devpar.ImgMakeborderWidth* i];
//		}
//	}
//	//���Ҷ�ͼת��Ϊ��ͼ ,��Χ�к�Բ�����궼��������img_out_rec����
//	cv::Mat img_out(Devpar.ImgHeight, Devpar.ImgMakeborderWidth, CV_8UC3);
//	cv::cvtColor(img_counter, img_out, cv::COLOR_GRAY2BGR);
//	//----------------------------------------------------------------------------------------------------------------------------------
//	//ɸѡ���
//	vector<CircleInfo>myInfo;
//	for (int j = 0; j < Devpar.ColThreadNum * Devpar.RowThreadNum; j++)
//	{
//		if (tHostIndex[j] != 0)
//		{
//			CircleInfo temp;
//			temp.index = (short)j;
//			temp.length = tHostLength[j];
//			temp.area = tHostArea[j];
//			temp.xpos = tHostXpos[j];
//			temp.ypos = tHostYpos[j];
//			myInfo.push_back(temp);
//			//���Ʒ�λ��
//			cv::Point  Rmin(tHostRecYLeft[j] - 1, tHostRecXLeft[j] - 1);
//			cv::Point  Rmax(tHostRecYRight[j] + 1, tHostRecXRight[j] + 1);
//			cv::rectangle(img_out, Rmin, Rmax, cv::Scalar(0, 0, 255));
//			img_out.at<cv::Vec3b>(temp.xpos, temp.ypos) = cv::Vec3b(0, 0, 255);
//		}
//	}
//	//д������
//	if (myInfo.size() > 0)
//	{
//		FILE* fp;
//		fp = fopen(strfilename, "wb");
//		fwrite(&myInfo[0], sizeof(CircleInfo)*myInfo.size(), 1, fp);
//		fclose(fp);
//	}
//	//��ȡ����������������Բ������
//	FILE *fr;
//	fr = fopen(strfilename, "rb");
//	if (fr != NULL)
//	{
//		fseek(fr, 0, SEEK_END);
//		long lSize = ftell(fr);
//		rewind(fr);
//		int num9 = lSize / sizeof(CircleInfo);
//		CircleInfo *RInfo = (CircleInfo*)malloc(sizeof(CircleInfo)*num9);
//		fread(RInfo, sizeof(CircleInfo), num9, fr);
//		//����ԭ��ͼ
//		for (int i = 0; i < num9; i++)
//		{
//			img_out.at<cv::Vec3b>(RInfo[i].xpos, RInfo[i].ypos) = cv::Vec3b(0, 255, 0);
//		}
//	}
//	fclose(fr);
//	//�ͷ��ڴ�
//	hipHostFree(tHostImage);
//	hipFree(tDevRecXLeft);
//	hipFree(tDevRecYLeft);
//	hipFree(tDevRecXRight);
//	hipFree(tDevRecYRight);
//	hipFree(tDevLength);
//	hipFree(tDevArea);
//	hipFree(tDevXpos);
//	hipFree(tDevYpos);
//	hipFree(tDevIndex);
//	hipFree(tDevImage);
//	hipFree(tDevpad);
//	hipFree(tDev2val);
//	hipFree(tDevcounter);
//	delete[]tHostRecXLeft;
//	delete[]tHostRecYLeft;
//	delete[]tHostRecXRight;
//	delete[] tHostRecYRight;
//	delete[]tHostLength;
//	delete[]tHostArea;
//	delete[]tHostXpos;
//	delete[]tHostYpos;
//	delete[]tHostIndex;
//	delete[]Src_counter;
//	return 0;
//}

