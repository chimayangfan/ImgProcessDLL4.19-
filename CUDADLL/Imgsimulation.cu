#include "hip/hip_runtime.h"
#include"Imgsimulation.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
#include <string.h>
#include <fstream> 
#include <string>
#include <io.h>
#include <vector>
#include <time.h>  

#include <stdio.h>  
#include<algorithm>

#include "Thread.h"
#include "ThreadPoolExecutor.h"
#include"hip/hip_runtime_api.h"
#include <hip/hip_runtime_api.h>//������
#include <Windows.h>
#include <GdiPlus.h>
#pragma comment( lib, "GdiPlus.lib" )
using namespace Gdiplus;
using namespace std;
using namespace cv;

//�����豸���ܶ���
#define CPUThreads 2
#define CUDAStreams 5
int gHostImgblock = CPUThreads * CUDAStreams;
int gDeviceCount;
int gHostPathImgNumber;
//����ͼƬ��С����block��thread���� 
#define gThreshold 60   //��ֵ������ֵ
#define counterNum 640  //����Ƕ������ȡ������Ϣʱ�����߳�����
#define gLengthMax 300//�ܳ������ֵ
#define gLengthMin 30 //�ܳ�����Сֵ 
const int gImgHeight = 5120;//����
const int gImgWidth = 5120; //����
const int gThreadNum = gImgHeight * gImgWidth / 64;

#define Pretreatment
#ifdef Pretreatment
#define ReadImageNumber 250
unsigned char* gHostImage[ReadImageNumber];
#endif // Pretreatment

unsigned char* rhost_in[CUDAStreams];//ҳ�����ڴ�
unsigned char* rDev_in[CUDAStreams];//�豸�ڴ�
unsigned char* rgpu_2val[CUDAStreams];//��ֵ��ͼ
unsigned char* rgpu_counter[CUDAStreams];//����ͼ����ִ��findcountores֮�������

unsigned char* shost_in[CUDAStreams];//ҳ�����ڴ�
unsigned char* sDev_in[CUDAStreams];//�豸�ڴ�
unsigned char* sgpu_2val[CUDAStreams];//��ֵ��ͼ
unsigned char* sgpu_counter[CUDAStreams];//����ͼ����ִ��findcountores֮�������

dim3 mGrid(20, 5120);//�����������Ϊȫ�ֱ���
dim3 mGrid2(5, 640);

hipStream_t *rcS;
hipStream_t *scS;

/*�ҶȻ��Ͷ�ֵ��*/
//�ú˺����߳�����λ  <<<(5,5120),1024>>>  ���ҵĵ���block�����1024, grid��һ��block����ͼ���һ�С�;   srcΪԭͼ��   dstΪ��ֵ��ͼ��  dst2Ϊ�Ҷ�ͼ
__global__ void Graybmp(unsigned char *src_gray, unsigned char *dst_2val, unsigned char *dst_counter)
{
	const int Id = threadIdx.x + (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x;//����άgrid��һάblock��
	int temp = int(src_gray[Id]);//�Ĵ����������أ���߷ô�Ч��								
	if (Id < gImgWidth*gImgHeight)
	{
		dst_2val[Id] = unsigned char(255 * int(temp>gThreshold));//��ֵ�������ü�������֧�ṹ
		dst_counter[Id] = unsigned char(255 * int(temp>gThreshold));
	}
}

/*��ȡ��������Ե��⣩*/
//����ʮ����4����ʴ��ԭͼ���ص���һ�������߳�����<<<(5,5120),1024>>>
//����-----�����ֵ��ͼ
__global__  void dilation(unsigned char *src, unsigned char *dst)
{
	const int Id_x = threadIdx.x + blockIdx.x *blockDim.x;//Id_x��������Ϣ  Id_y��������Ϣ
	const int Id_y = blockIdx.y;//Id_y��������Ϣ
	int temp;
	if (Id_x > 1 && Id_x < (gImgWidth - 2) && Id_y>1 && Id_y < gridDim.y - 1)
	{
		if (src[Id_x + Id_y * gImgWidth] == 0)
		{
			temp = int(src[Id_x - 1 + (Id_y - 1)*gImgWidth]) + int(src[Id_x + (Id_y - 1)*gImgWidth]) + int(src[Id_x + 1 + (Id_y - 1)*gImgWidth])
				+ int(src[Id_x - 1 + Id_y * gImgWidth]) + int(src[Id_x + 1 + Id_y * gImgWidth]) +
				int(src[Id_x - 1 + (Id_y + 1)*gImgWidth]) + int(src[Id_x + (Id_y + 1)*gImgWidth]) + int(src[Id_x + 1 + (Id_y + 1)*gImgWidth]);//��4��������
			dst[Id_x + Id_y * gImgWidth] = temp > 0 ? 255 : 0;
		}
	}

}


//��ʴ
__global__  void erosion(unsigned char *src, unsigned char *dst)
{
	const int Id_x = threadIdx.x + blockIdx.x *blockDim.x;//Id_x��������Ϣ  Id_y��������Ϣ
	const int Id_y = blockIdx.y;//Id_y��������Ϣ
	int temp;
	//����4����ֵ�Ϳ��ڲ��㣬��ȡ������Ϣ�����ڵ�dst���Ǵ洢��������Ϣ
	if (Id_x > 0 && Id_x < (gImgWidth - 1) && Id_y>0 && Id_y < gridDim.y)
	{
		if (src[Id_x + Id_y * gImgWidth] != 0)
		{
			temp = int(src[Id_x + (Id_y - 1)*gImgWidth]) + int(src[Id_x - 1 + Id_y * gImgWidth]) +
				int(src[Id_x + 1 + Id_y * gImgWidth]) + int(src[Id_x + (Id_y + 1)*gImgWidth]);//��4����ʴ
			dst[Id_x + Id_y * gImgWidth] = temp >= 1020 ? 0 : 255;
		}
	}
}


/*������ȡ*/
//���ð�����׷�ٷ���ȡ�������߳�����Ϊ<<<640��640>>>��   һ���̴߳���16*16��С������������ȡ׼�������±߽�����ԭ�� 
//srcΪ�������飨��Ե���������c_length��ȡ���ܳ�ֵ��   (x_min,y_min)��(x_max,y_max)���ڱ���������������,����x�йص�Ϊ��������y�йصı���������
__global__  void getCounter(unsigned char *src, short *c_length, short* x_min, short * y_min, short* x_max, short *y_max)
{
	//�����������飬���ڸ���������,��ʼ������Ϊ���ҷ���0��λ����˳ʱ����ת45�㣨������1��
	const  int direction_y[8] = { 1,1,0,-1,-1,-1,0,1 };
	const  int direction_x[8] = { 0,1,1,1,0,-1,-1,-1 };
	//��ȡ����������
	const int y = (blockIdx.x*blockDim.x + threadIdx.x) * 8;//y��������
	const int x = blockIdx.y * 8;//x��������
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	/*��ʼ��������ֵ*/
	c_length[Id] = 0;
	x_min[Id] = 0;
	x_max[Id] = 0;
	y_min[Id] = 0;
	y_max[Id] = 0;

	/*��ʼ������������Ϣ*/
	short x_pos_max = 0, x_pos_min = 0;
	short y_pos_max = 0, y_pos_min = 0;//����������������λ����Ϣ
	short  Point_counts = 0;//��������

							/*ѭ����ȡ�����ܳ���Ϣ*/
	for (int i = x; i < (x + 8); i++)
	{
		for (int j = y; j < (y + 8); j++)
		{
			if (255 == src[j + i * gImgWidth])
			{
				//���±߽�����
				y_pos_max = j;
				y_pos_min = j;
				x_pos_max = i;
				x_pos_min = i;
				Point_counts = 1;//������Ŀ����ֵ
								 // ��ʼ�㼰��ǰ��  
				short x_pos = i;//������
				short y_pos = j;//������
				bool first_time = true;//�Ƿ�ʱ��һ�λ�ȡ������
				short counts = 0;//����ѭ������
				short curr_d = 0;//������������������ȡֵ0-7��ʾ�������8�����õķ�λ
								 // ���и���  
				for (short cLengthCount = 0; cLengthCount<gLengthMax; cLengthCount++)//�����ѭ��������Ҫ�ý������õ��ܳ����ֵ��ȷ��
				{
					//�������ǵ�
					short root_x = x_pos;
					short root_y = y_pos;

					//������������߽�����
					x_pos_max = x_pos_max > x_pos ? x_pos_max : x_pos;
					y_pos_max = y_pos_max > y_pos ? y_pos_max : y_pos;
					x_pos_min = x_pos_min < x_pos ? x_pos_min : x_pos;
					y_pos_min = y_pos_min < y_pos ? y_pos_min : y_pos;


					// ѭ���˴� :���ڻ�ȡ��һ��������
					for (counts = 0; counts < 8; counts++)
					{
						// ��ֹ��������  
						curr_d -= curr_d >= 8 ? 8 : 0;
						curr_d += curr_d < 0 ? 8 : 0;

						//��ʵ�ϣ�ֻ��Ҫ�ж�7�������ڵ���Ϣ(���˵�һ��֮��)
						if (first_time && (counts == 6))
						{
							first_time = false;
							continue;
						}
						//���±�ǵ�root;

						root_x = x_pos + direction_x[curr_d];//����������
						root_y = y_pos + direction_y[curr_d];//����������

															 //�жϵ��Ƿ�Խ�磬����ͼ�����������
						if (root_x < 0 || root_x >= gImgHeight || root_y < 0 || root_y >= gImgWidth)
						{
							curr_d++;
							continue;
						}
						//������ڱ�Ե  
						if (255 == src[root_y + root_x * gImgWidth])
						{
							curr_d -= 2;   //���µ�ǰ����  
							Point_counts++;
							//����b_pt:���ٵ�root��  
							x_pos = root_x;
							y_pos = root_y;
							break;   // ����forѭ��  
						}
						curr_d++;
					}   // end for  

						//���ٽ��������쳣����
					if (8 == counts || (x_pos >= (x + 8) && y_pos >= (y + 8)))
					{
						break;
					}
					//��������
					if (y_pos == j && x_pos == i)
					{
						//����������Ϣ
						c_length[Id] = Point_counts;
						x_min[Id] = x_pos_min;
						x_max[Id] = x_pos_max;
						y_min[Id] = y_pos_min;
						y_max[Id] = y_pos_max;
						break;
					}//��������if
				}//��Χfor����			
			}//�ж�ǰ����if����
			j = y_pos_max>j ? y_pos_max : j;//���º�����������
		}//��һ��for����
		i = x_pos_max>i ? x_pos_max : i;
	}//�ڶ���for ����
}//�˺�������

 //��������getCounter��ȡ�� x_min��x_max�����У� y_min��y_max�����С�
 //����˺�����getCounter���õ��߳�������<<<640,640>>>�����ڷ��ص�ַ�����������
__global__  void getInfo(unsigned char* src_gray, unsigned char* src_counter, short *length, short* x_min, short * y_min, short* x_max, short *y_max, short *xpos, short*ypos, short *area)
{
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	short count = 0;//�����������
	int sum_gray = 0;//Բ������ĻҶ�ֵ֮��
	int x_sum = 0;//x�Ҷ�ֵ��Ȩ��
	int y_sum = 0;//y�Ҷ�ֵ��Ȩ��
				  //����ÿ��ѭ���ж϶�Ҫ���ʱ߽磬���Ը��üĴ����洢�߽硣
	short xmm = x_min[Id];
	short xmx = x_max[Id];
	short ymm = y_min[Id];
	short ymx = y_max[Id];
	short jcount = ((ymx - ymm) / 4 + 1) * 4;
	unsigned char temp0, temp1, temp2, temp3;//�üĴ����ݴ�ͼ�����ݣ���Сȫ���ڴ�ķ��ʣ���߷ô�Ч��
	area[Id] = 0;
	xpos[Id] = 0;
	ypos[Id] = 0;
	if (length[Id] > gLengthMin)
	{
		//ѭ���Ż�,�������������һЩ�����ֵ����Ҫ����һ�£�
		for (int i = xmm; i <= xmx; i++)
			for (int j = ymm; j <= ymm + jcount; j = j + 4)
			{
				//��ֹԽ��
				temp0 = j > ymx ? 0 : 1;
				temp1 = j > ymx ? 0 : 1;
				temp2 = j > ymx ? 0 : 1;
				temp3 = j > ymx ? 0 : 1;


				temp0 *= src_gray[j + i * gImgWidth];
				temp1 *= src_gray[j + 1 + i * gImgWidth];
				temp2 *= src_gray[j + 2 + i * gImgWidth];
				temp3 *= src_gray[j + 3 + i * gImgWidth];

				count += temp0>0 ? 1 : 0; //�������
				count += temp1>0 ? 1 : 0;
				count += temp2>0 ? 1 : 0;
				count += temp3>0 ? 1 : 0;

				sum_gray += temp0 + temp1 + temp2 + temp3;


				x_sum += i * temp0 + i * temp1 + i * temp2 + i * temp3;
				y_sum += j * temp0 + (j + 1)*temp1 + (j + 2)*temp2 + (j + 3)*temp3;
			}
		area[Id] = count;
		xpos[Id] = x_sum / sum_gray;
		ypos[Id] = y_sum / sum_gray;
	}
}

//ɸѡ���ظ���Ϣ�ĺ���,����˺���Ҫ����ʧ��
__global__  void getTrueInfo(short *xcenter, short *ycenter, short*index)
{
	const int Id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	short temp = 0;
	index[Id] = 0;//����������
	if ((Id > counterNum) && (Id < counterNum*(counterNum - 1)))
	{
		if (xcenter[Id] != 0)
		{
			//�ж�һ���̻߳�ȡ�������Ƿ���������ڵ��ҷ��̣߳���+1�����·��̣߳���+1����ȡ������һ�¡�����һ�����������ֵ
			//��
			temp += ((xcenter[Id] == xcenter[Id + 1]) && (ycenter[Id] == ycenter[Id + 1])) ? 1 : 0;//��
			temp += ((xcenter[Id] == xcenter[Id + 640]) && (ycenter[Id] == ycenter[Id + 640])) ? 1 : 0;//��
			temp += ((xcenter[Id] == xcenter[Id - 639]) && (ycenter[Id] == ycenter[Id - 639])) ? 1 : 0;//����
			index[Id] = temp > 0 ? 0 : Id;
		}
	}
	//�¼ӵ�����ѡȡģʽ
}

void GPUDeviceCheck()
{
	hipError_t hipGetDeviceCount(int* count);
	hipGetDeviceCount(&gDeviceCount);
	for (int i = 0; i<gDeviceCount; i++)
	{
		hipDeviceProp_t DevProp;
		hipGetDeviceProperties(&DevProp, i);
		printf("Device %d has compute capability %d.%d \n", i, DevProp.major, DevProp.minor);
	}
}

//8λ�Ҷ�BMP��ʽͼ���ȡ
unsigned char *RmwRead8BitBmpFile2Img(const char * filename, int *width, int *height) {
	FILE *binFile;
	unsigned char *pImg = NULL;
	BITMAPFILEHEADER fileHeader;
	BITMAPINFOHEADER bmpHeader;
	BOOL isRead = TRUE;
	int linenum, ex; //linenum:һ�����ص��ֽ���������������ֽ� 

					 //open file
	if ((binFile = fopen(filename, "rb")) == NULL) return NULL;

	//read struts
	if (fread((void *)&fileHeader, 1, sizeof(fileHeader), binFile) != sizeof(fileHeader)) isRead = FALSE;
	if (fread((void *)&bmpHeader, 1, sizeof(bmpHeader), binFile) != sizeof(bmpHeader)) isRead = FALSE;


	if (isRead == FALSE || fileHeader.bfOffBits<sizeof(fileHeader) + sizeof(bmpHeader)) {
		fclose(binFile);
		return NULL;
	}

	//read image info
	*width = bmpHeader.biWidth;
	*height = bmpHeader.biHeight;
	linenum = (*width * 1 + 3) / 4 * 4;
	ex = linenum - *width * 1;         //ÿһ�е�����ֽ�

	fseek(binFile, fileHeader.bfOffBits, SEEK_SET);
	pImg = new unsigned char[(*width)*(*height)];
	if (pImg != NULL) {
		for (int i = 0; i<*height; i++) {
			int r = fread(pImg + (*height - i - 1)*(*width), sizeof(unsigned char), *width, binFile);
			if (r != *width) {
				delete pImg;
				fclose(binFile);
				return NULL;
			}
			fseek(binFile, ex, SEEK_CUR);
		}
	}
	fclose(binFile);
	return pImg;
}

//BMP��ʽͼ��д��
bool RmwWrite8bitImg2BmpFile(unsigned char *pImg, int width, int height, const char * filename)
{
	FILE * BinFile;
	BITMAPFILEHEADER FileHeader;
	BITMAPINFOHEADER BmpHeader;
	int i, extend;
	bool Suc = true;
	unsigned char p[4], *pCur;
	unsigned char* ex;

	extend = (width + 3) / 4 * 4 - width;

	// Open File
	if ((BinFile = fopen(filename, "w+b")) == NULL) { return false; }
	//��������ṹ����
	FileHeader.bfType = ((WORD)('M' << 8) | 'B');
	FileHeader.bfOffBits = sizeof(BITMAPFILEHEADER) + sizeof(BITMAPINFOHEADER) + 256 * 4L;//2��ͷ�ṹ��ӵ�ɫ��
	FileHeader.bfSize = FileHeader.bfOffBits + (width + extend)*height;
	FileHeader.bfReserved1 = 0;
	FileHeader.bfReserved2 = 0;
	if (fwrite((void *)&FileHeader, 1, sizeof(FileHeader), BinFile) != sizeof(FileHeader)) Suc = false;
	// Fill the ImgHeader
	BmpHeader.biSize = 40;
	BmpHeader.biWidth = width;
	BmpHeader.biHeight = height;
	BmpHeader.biPlanes = 1;
	BmpHeader.biBitCount = 8;
	BmpHeader.biCompression = 0;
	BmpHeader.biSizeImage = 0;
	BmpHeader.biXPelsPerMeter = 0;
	BmpHeader.biYPelsPerMeter = 0;
	BmpHeader.biClrUsed = 0;
	BmpHeader.biClrImportant = 0;
	if (fwrite((void *)&BmpHeader, 1, sizeof(BmpHeader), BinFile) != sizeof(BmpHeader)) Suc = false;
	// д���ɫ��
	for (i = 0, p[3] = 0; i<256; i++)
	{
		p[0] = p[1] = p[2] = i; // blue,green,red; //��255 - i��Ҷȷ�ת
		if (fwrite((void *)p, 1, 4, BinFile) != 4) { Suc = false; break; }
	}

	if (extend)
	{
		ex = new unsigned char[extend]; //��������СΪ 0~3
		memset(ex, 0, extend);
	}

	//write data
	for (pCur = pImg + (height - 1)*width; pCur >= pImg; pCur -= width)
	{
		if (fwrite((void *)pCur, 1, width, BinFile) != (unsigned int)width) Suc = false; // ��ʵ������
		if (extend) // ��������� �������0
			if (fwrite((void *)ex, 1, extend, BinFile) != 1) Suc = false;
	}

	// return;
	fclose(BinFile);
	if (extend)
		delete[] ex;
	return Suc;
}

class R : public Runnable
{
public:
	~R()
	{
	}
	void Run()
	{
		//������
		hipError_t  err;
		int img_index = 0;
		int Width;
		int Height;
		char strFilename[100];                                          //��1������һ���ַ����鱣��----ͼƬ�Ķ�ȡ·�� 
		char saveFilename[100];                                         //��1������һ���ַ����鱣��----ͼƬ�Ĵ洢·��
		char* path = "C:\\pic\\img_data";

		//����GPU�豸��
		hipSetDevice(0);
		/*������*/
		//����
		short *gpHostLength[CUDAStreams];
		short *gpHostArea[CUDAStreams];
		short *gpHostXpos[CUDAStreams];
		short *gpHostYpos[CUDAStreams];
		short *gpHostIndex[CUDAStreams];
		/*�豸��*/
		short *  gpDevRecXLeft[CUDAStreams];
		short *  gpDevRecYLeft[CUDAStreams];
		short *  gpDevRecXRight[CUDAStreams];
		short *  gpDevRecYRight[CUDAStreams];
		//���
		short  *gpDevLength[CUDAStreams];
		short  *gpDevArea[CUDAStreams];
		short  *gpDevXpos[CUDAStreams];
		short  *gpDevYpos[CUDAStreams];
		short  *gpDevIndex[CUDAStreams];

		for (int i = 0; i < CUDAStreams; i++)
		{
			checkCudaErrors(hipHostAlloc((void**)&gpHostLength[i], gThreadNum * sizeof(short), hipHostMallocDefault));//����ܳ�
			checkCudaErrors(hipHostAlloc((void**)&gpHostArea[i], gThreadNum * sizeof(short), hipHostMallocDefault));//���
			checkCudaErrors(hipHostAlloc((void**)&gpHostXpos[i], gThreadNum * sizeof(short), hipHostMallocDefault));//��������x
			checkCudaErrors(hipHostAlloc((void**)&gpHostYpos[i], gThreadNum * sizeof(short), hipHostMallocDefault));//��������y
			checkCudaErrors(hipHostAlloc((void**)&gpHostIndex[i], gThreadNum * sizeof(short), hipHostMallocDefault));//����������
			checkCudaErrors(hipMalloc((void**)&gpDevRecXLeft[i], gThreadNum * sizeof(short)));//��λ�� xmin
			checkCudaErrors(hipMalloc((void**)&gpDevRecYLeft[i], gThreadNum * sizeof(short)));//	    ymin
			checkCudaErrors(hipMalloc((void**)&gpDevRecXRight[i], gThreadNum * sizeof(short)));//		xmax
			checkCudaErrors(hipMalloc((void**)&gpDevRecYRight[i], gThreadNum * sizeof(short)));//		ymax
			checkCudaErrors(hipMalloc((void**)&gpDevLength[i], gThreadNum * sizeof(short)));//�豸�����	�ܳ�
			checkCudaErrors(hipMalloc((void**)&gpDevArea[i], gThreadNum * sizeof(short)));//				���
			checkCudaErrors(hipMalloc((void**)&gpDevXpos[i], gThreadNum * sizeof(short)));//				xpos
			checkCudaErrors(hipMalloc((void**)&gpDevYpos[i], gThreadNum * sizeof(short)));//				ypos
			checkCudaErrors(hipMalloc((void**)&gpDevIndex[i], gThreadNum * sizeof(short)));//				������
		}

		while ((img_index + CUDAStreams) <= gHostPathImgNumber)
		{
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(rDev_in[i], gHostImage[img_index + i], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyHostToDevice, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				Graybmp << <mGrid, 256, 0, rcS[i] >> > (rDev_in[i], rgpu_2val[i], rgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//�߽���ȡ
				dilation << <mGrid, 256, 0, rcS[i] >> >(rgpu_2val[i], rgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(rgpu_2val[i], rgpu_counter[i], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyDeviceToDevice, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				erosion << <mGrid, 256, 0, rcS[i] >> > (rgpu_2val[i], rgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ�����ͱ�Ե��
				getCounter << <mGrid2, 128, 0, rcS[i] >> > (rgpu_counter[i], gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i]);//��ȡ�����ĺ���
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ���������//��ȡ������Ϣ�˺���
				getInfo << <mGrid2, 128, 0, rcS[i] >> > (rDev_in[i], rgpu_counter[i], gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i], gpDevXpos[i], gpDevYpos[i], gpDevArea[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ɸѡ��ȡ������������ķ��ظ���Ϣ
				getTrueInfo << <mGrid2, 128, 0, rcS[i] >> > (gpDevXpos[i], gpDevYpos[i], gpDevIndex[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostLength[i], gpDevLength[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostArea[i], gpDevArea[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostXpos[i], gpDevXpos[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostYpos[i], gpDevYpos[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostIndex[i], gpDevIndex[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, rcS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				FILE* fp;
				sprintf_s(strFilename, "%s\\%d.bin", path, img_index + i + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ�
				fp = fopen(strFilename, "wb");
				fwrite(gpHostLength[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostArea[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostXpos[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostYpos[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostIndex[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fclose(fp);
			}
			img_index += gHostImgblock;
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipStreamSynchronize(rcS[i]);
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipHostFree(gpHostLength[i]);
			hipHostFree(gpHostArea[i]);
			hipHostFree(gpHostXpos[i]);
			hipHostFree(gpHostYpos[i]);
			hipHostFree(gpHostIndex[i]);
			//�豸���ڴ�
			hipFree(gpDevRecXLeft[i]);
			hipFree(gpDevRecYLeft[i]);
			hipFree(gpDevRecXRight[i]);
			hipFree(gpDevRecYRight[i]);
			hipFree(gpDevLength[i]);
			hipFree(gpDevArea[i]);
			hipFree(gpDevXpos[i]);
			hipFree(gpDevYpos[i]);
			hipFree(gpDevIndex[i]);
		}
	}
};

class S : public Runnable
{
public:
	~S()
	{
	}
	void Run()
	{
		int img_index = 0;
		int Width;
		int Height;
		char strFilename[100];                                          //��1������һ���ַ����鱣��----ͼƬ�Ķ�ȡ·�� 
		char saveFilename[100];                                         //��1������һ���ַ����鱣��----ͼƬ�Ĵ洢·��
		char* path = "C:\\pic\\img_data";

		//����GPU�豸��
		hipSetDevice(1);
		/*������*/
		//����
		short *gpHostLength[CUDAStreams];
		short *gpHostArea[CUDAStreams];
		short *gpHostXpos[CUDAStreams];
		short *gpHostYpos[CUDAStreams];
		short *gpHostIndex[CUDAStreams];
		/*//������ʾ�ô��룬���Գɹ������Ρ�������
		//��ʱ�����������˰�Χ��
		short *rec_xmin[CUDAStreams];
		short *rec_ymin[CUDAStreams];
		short *rec_xmax[CUDAStreams];
		short *rec_ymax[CUDAStreams];
		/*������ʾ�ô��룬���Գɹ������Ρ�������*/
		/*�豸��*/
		short *gpDevRecXLeft[CUDAStreams];
		short *gpDevRecYLeft[CUDAStreams];
		short *gpDevRecXRight[CUDAStreams];
		short *gpDevRecYRight[CUDAStreams];
		//���
		short  *gpDevLength[CUDAStreams];
		short  *gpDevArea[CUDAStreams];
		short  *gpDevXpos[CUDAStreams];
		short  *gpDevYpos[CUDAStreams];
		short  *gpDevIndex[CUDAStreams];

		for (int i = 0; i < CUDAStreams; i++)
		{
			checkCudaErrors(hipHostAlloc((void**)&gpHostLength[i], gThreadNum * sizeof(short), hipHostMallocDefault));//����ܳ�
			checkCudaErrors(hipHostAlloc((void**)&gpHostArea[i], gThreadNum * sizeof(short), hipHostMallocDefault));//���
			checkCudaErrors(hipHostAlloc((void**)&gpHostXpos[i], gThreadNum * sizeof(short), hipHostMallocDefault));//��������x
			checkCudaErrors(hipHostAlloc((void**)&gpHostYpos[i], gThreadNum * sizeof(short), hipHostMallocDefault));//��������y
			checkCudaErrors(hipHostAlloc((void**)&gpHostIndex[i], gThreadNum * sizeof(short), hipHostMallocDefault));//����������
			/*//������ʾ�ô��룬���Գɹ������Ρ�������
			//��ʱ�����������˰�Χ��
			checkCudaErrors(hipHostAlloc((void**)&rec_xmin[i], gThreadNum * sizeof(short), hipHostMallocDefault));//xmin
			checkCudaErrors(hipHostAlloc((void**)&rec_ymin[i], gThreadNum * sizeof(short), hipHostMallocDefault));//ymin
			checkCudaErrors(hipHostAlloc((void**)&rec_xmax[i], gThreadNum * sizeof(short), hipHostMallocDefault));//xmax
			checkCudaErrors(hipHostAlloc((void**)&rec_ymax[i], gThreadNum * sizeof(short), hipHostMallocDefault));//ymax
			/*������ʾ�ô��룬���Գɹ������Ρ�������*/
			checkCudaErrors(hipMalloc((void**)&gpDevRecXLeft[i], gThreadNum * sizeof(short)));//��λ�� xmin
			checkCudaErrors(hipMalloc((void**)&gpDevRecYLeft[i], gThreadNum * sizeof(short)));//	    ymin
			checkCudaErrors(hipMalloc((void**)&gpDevRecXRight[i], gThreadNum * sizeof(short)));//		xmax
			checkCudaErrors(hipMalloc((void**)&gpDevRecYRight[i], gThreadNum * sizeof(short)));//		ymax
			checkCudaErrors(hipMalloc((void**)&gpDevLength[i], gThreadNum * sizeof(short)));//�豸�����	�ܳ�
			checkCudaErrors(hipMalloc((void**)&gpDevArea[i], gThreadNum * sizeof(short)));//				���
			checkCudaErrors(hipMalloc((void**)&gpDevXpos[i], gThreadNum * sizeof(short)));//				xpos
			checkCudaErrors(hipMalloc((void**)&gpDevYpos[i], gThreadNum * sizeof(short)));//				ypos
			checkCudaErrors(hipMalloc((void**)&gpDevIndex[i], gThreadNum * sizeof(short)));//				������
		}

		while ((img_index + CUDAStreams * 2) <= gHostPathImgNumber)
		{
			for (int i = 0; i < CUDAStreams; i++)
			{
				//hipMemcpyAsync(sDev_in[i], gHostImage[img_index + i + CUDAStreams], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyHostToDevice, scS[i]);
				hipMemcpy(sDev_in[i], gHostImage[img_index + i + CUDAStreams], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyHostToDevice);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ִ�лҶȻ�����ֵ���˺�������
				Graybmp << <mGrid, 256, 0, scS[i] >> > (sDev_in[i], sgpu_2val[i], sgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//�߽���ȡ
				dilation << <mGrid, 256, 0, scS[i] >> >(sgpu_2val[i], sgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(sgpu_2val[i], sgpu_counter[i], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyDeviceToDevice, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				erosion << <mGrid, 256, 0, scS[i] >> > (sgpu_2val[i], sgpu_counter[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ�����ͱ�Ե��
				getCounter << <mGrid2, 128, 0, scS[i] >> > (sgpu_counter[i], gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i]);//��ȡ�����ĺ���
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//��ȡ���������//��ȡ������Ϣ�˺���
				getInfo << <mGrid2, 128, 0, scS[i] >> > (sDev_in[i], sgpu_counter[i], gpDevLength[i], gpDevRecXLeft[i], gpDevRecXRight[i], gpDevRecYLeft[i], gpDevRecYRight[i], gpDevXpos[i], gpDevYpos[i], gpDevArea[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				//ɸѡ��ȡ������������ķ��ظ���Ϣ
				getTrueInfo << <mGrid2, 128, 0, scS[i] >> > (gpDevXpos[i], gpDevYpos[i], gpDevIndex[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostLength[i], gpDevLength[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostArea[i], gpDevArea[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostXpos[i], gpDevXpos[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostYpos[i], gpDevYpos[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
				hipMemcpyAsync(gpHostIndex[i], gpDevIndex[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			/*//������ʾ�ô��룬���Գɹ������Ρ�������
			for (int i = 0; i < CUDAStreams; i++)
			{
			hipMemcpyAsync(rec_xmin[i], gpDevRecXLeft[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
			hipMemcpyAsync(rec_ymin[i], gpDevRecXRight[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
			hipMemcpyAsync(rec_xmax[i], gpDevRecYLeft[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}
			for (int i = 0; i < CUDAStreams; i++)
			{
			hipMemcpyAsync(rec_ymax[i], gpDevRecYRight[i], sizeof(short)* counterNum * counterNum, hipMemcpyDeviceToHost, scS[i]);
			}

			for (int i = 0; i < CUDAStreams; i++)
			{
			//����1��
			uchar *img_counter = new uchar[gImgHeight *gImgWidth];
			hipMemcpy(img_counter, sgpu_counter[i], sizeof(uchar)* gImgHeight *gImgWidth, hipMemcpyDeviceToHost);

			//printf("%s", hipGetErrorString(err));
			Mat img_out_counter(gImgWidth, gImgHeight , CV_8UC1);
			for (int j = 0; j < gImgWidth; j++)
			{
			uchar* data = img_out_counter.ptr<uchar>(j);  //��ȡ��i�е��׵�ַ��
			for (int k = 0; k < gImgHeight ; k++)   //��ѭ��
			{
			data[k] = img_counter[k + j * gImgHeight ];
			}
			}
			//����2-��ǵ��Ƿ���ȡ��ȷ��
			Mat img_out_rect(gImgWidth, gImgHeight , CV_8UC3);
			cvtColor(img_out_counter, img_out_rect, COLOR_GRAY2BGR);

			//ɸѡ��ӡ��ȡ������
			vector<CircleInfo>myInfo;
			for (int j = 0; j < counterNum * counterNum; j++)
			{
			if (gpHostIndex[i][j] != 0)
			{
			CircleInfo temp;
			temp.xpos = gpHostXpos[i][j];
			temp.ypos = gpHostYpos[i][j];
			temp.cLength = gpHostLength[i][j];
			temp.sArea = gpHostArea[i][j];
			myInfo.push_back(temp);
			//������Χ�о��ο�
			cv::Point temptop(rec_ymin[i][j] - 1, rec_xmin[i][j] - 1);
			cv::Point tempdown(rec_ymax[i][j] + 1, rec_xmax[i][j] + 1);
			rectangle(img_out_rect, temptop, tempdown, Scalar(0, 0, 255), 1, 1, 0);
			img_out_rect.at<Vec3b>(gpHostXpos[i][j], gpHostYpos[i][j])[2] = 255;
			}

			}
			//cout << endl << "��������Ŀ" << myInfo.size() << endl;

			//sprintf_s(strFilename, "C:\\pic\\img_write\\%d.bmp", img_index + i + CUDAStreams + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ�
			//imwrite(strFilename, img_out_counter);
			delete[]img_counter;
			}
			/*������ʾ�ô��룬���Գɹ������Ρ�������*/
			for (int i = 0; i < CUDAStreams; i++)
			{
				FILE* fp;
				sprintf_s(strFilename, "%s\\%d.bin", path, img_index + i + CUDAStreams + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ�
				fp = fopen(strFilename, "wb");
				fwrite(gpHostLength[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostArea[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostXpos[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostYpos[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fwrite(gpHostIndex[i], 1, sizeof(short)* counterNum * counterNum, fp);
				fclose(fp);
			}
			img_index += gHostImgblock;
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipStreamSynchronize(scS[i]);
		}
		for (int i = 0; i < CUDAStreams; i++)
		{
			hipHostFree(gpHostLength[i]);
			hipHostFree(gpHostArea[i]);
			hipHostFree(gpHostXpos[i]);
			hipHostFree(gpHostYpos[i]);
			hipHostFree(gpHostIndex[i]);
			/*//������ʾ�ô��룬���Գɹ������Ρ�������
			hipHostFree(rec_xmin[i]);
			hipHostFree(rec_ymin[i]);
			hipHostFree(rec_xmax[i]);
			hipHostFree(rec_ymax[i]);
			/*������ʾ�ô��룬���Գɹ������Ρ�������*/
			//�豸���ڴ�
			hipFree(gpDevRecXLeft[i]);
			hipFree(gpDevRecYLeft[i]);
			hipFree(gpDevRecXRight[i]);
			hipFree(gpDevRecYRight[i]);
			hipFree(gpDevLength[i]);
			hipFree(gpDevArea[i]);
			hipFree(gpDevXpos[i]);
			hipFree(gpDevYpos[i]);
			hipFree(gpDevIndex[i]);
		}
	}
};
/*����ӿں���*/

//��������
//ͼ��·������ʽ���
IMGSIMULATION_API void Image_path_check(const char *path, const char *exten)
{
	Directory dir;
	string filepath(path);
	string fileexten(exten);

	vector<string> filenames = dir.GetListFiles(filepath, fileexten, false);

	if (filenames.size() == NULL)
	{
		perror(" There is no .BMP file! ");
		exit(0);
	}
	else
	{
		gHostPathImgNumber = filenames.size();
	}
	//ͼ��Ԥ������Ӳ�����������ڴ�
	#ifdef Pretreatment
		char strFilename[100];
		int mWidth;
		int mHeight;
		for (int i = 0; i < ReadImageNumber; i++)
		{
			sprintf_s(strFilename, "%s\\%d.bmp", path, i + 1); //��3����ͼƬ��·������̬��д�뵽strFilename�����ַ���ڴ�ռ� 
			checkCudaErrors(hipHostAlloc((void**)&gHostImage[i], gImgHeight * gImgWidth * sizeof(unsigned char), hipHostMallocDefault));
			gHostImage[i] = RmwRead8BitBmpFile2Img(strFilename, &mWidth, &mHeight);
		}
	#endif // Pretreatment
}

//����ԭͼ�������
IMGSIMULATION_API void SimulationImageTest(const char *path, int mWidth, int mHeight)
{
	char strFilename[150];
	for (int i = 0; i < 100; i++)
	{
		checkCudaErrors(hipHostAlloc((void**)&gHostImage[i], gImgHeight * gImgWidth * sizeof(unsigned char), hipHostMallocDefault));
		gHostImage[i] = RmwRead8BitBmpFile2Img(path, &mWidth, &mHeight);
	}
	gHostPathImgNumber = 100;
	CThreadPoolExecutor * pExecutor = new CThreadPoolExecutor();
	pExecutor->Init(1, CPUThreads, 1);
	R r;
	S s;

	pExecutor->Execute(&r, 0x01);
	pExecutor->Execute(&s, 0x02);

	pExecutor->Terminate();
	delete pExecutor;
	//�ͷ��ڴ�
	for (int i = 0; i < 100; i++)
	{
		hipHostFree(gHostImage[i]);
	}
}

//ȫ���ڴ�����
IMGSIMULATION_API void Memory_application()
{
	hipSetDevice(0);
	rcS = (hipStream_t *)malloc(CUDAStreams * sizeof(hipStream_t));
	hipSetDevice(1);
	scS = (hipStream_t *)malloc(CUDAStreams * sizeof(hipStream_t));
	for (int i = 0; i < CUDAStreams; i++)
	{
		hipSetDevice(0);
		checkCudaErrors(hipStreamCreate(&(rcS[i])));
		//checkCudaErrors(hipHostAlloc((void**)&rhost_in[i], gImgHeight * gImgWidth * sizeof(unsigned char), hipHostMallocDefault));
		checkCudaErrors(hipMalloc((void**)&rDev_in[i], gImgHeight * gImgWidth * sizeof(unsigned char)));
		hipMalloc((void**)&rgpu_2val[i], sizeof(unsigned char)*gImgHeight*gImgWidth);
		hipMalloc((void**)&rgpu_counter[i], sizeof(unsigned char)*gImgHeight*gImgWidth);

		hipSetDevice(1);
		checkCudaErrors(hipStreamCreate(&(scS[i])));
		//checkCudaErrors(hipHostAlloc((void**)&shost_in[i], gImgHeight * gImgWidth * sizeof(unsigned char), hipHostMallocDefault));
		checkCudaErrors(hipMalloc((void**)&sDev_in[i], gImgHeight * gImgWidth * sizeof(unsigned char)));
		hipMalloc((void**)&sgpu_2val[i], sizeof(unsigned char)*gImgHeight*gImgWidth);
		hipMalloc((void**)&sgpu_counter[i], sizeof(unsigned char)*gImgHeight*gImgWidth);
	}

}

//ȫ���ڴ��ͷ�
IMGSIMULATION_API void Memory_release()
{
	#ifdef Pretreatment
		for (int i = 0; i < ReadImageNumber; i++)
		{
			hipHostFree(gHostImage[i]);
		}
	#endif // Pretreatment
	for (int i = 0; i<CUDAStreams; i++)
	{
		hipSetDevice(0);
		//hipHostFree(rhost_in[i]);
		hipFree(rDev_in[i]);
		hipFree(rgpu_2val[i]);
		hipFree(rgpu_counter[i]);
		checkCudaErrors(hipStreamDestroy(rcS[i]));

		hipSetDevice(1);
		//hipHostFree(shost_in[i]);
		hipFree(sDev_in[i]);
		hipFree(sgpu_2val[i]);
		hipFree(sgpu_counter[i]);
		checkCudaErrors(hipStreamDestroy(scS[i]));
	}
}